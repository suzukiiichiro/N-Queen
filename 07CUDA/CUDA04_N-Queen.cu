/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc -O3 CUDA**_N-Queen.cu && ./a.out (-c|-r|-g|-s)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 
                    -s SGPU(サマーズ版と思われる)

 ４．バックトラック＋対称解除法

 　一つの解には、盤面を９０度、１８０度、２７０度回転、及びそれらの鏡像の合計
 　８個の対称解が存在する。対照的な解を除去し、ユニーク解から解を求める手法。
 
 ■ユニーク解の判定方法
   全探索によって得られたある１つの解が、回転・反転などによる本質的に変わること
 のない変換によって他の解と同型となるものが存在する場合、それを別の解とはしない
 とする解の数え方で得られる解を「ユニーク解」といいます。つまり、ユニーク解とは、
 全解の中から回転・反転などによる変換によって同型になるもの同士をグループ化する
 ことを意味しています。
 
   従って、ユニーク解はその「個数のみ」に着目され、この解はユニーク解であり、こ
 の解はユニーク解ではないという定まった判定方法はありません。ユニーク解であるか
 どうかの判断はユニーク解の個数を数える目的の為だけに各個人が自由に定義すること
 になります。もちろん、どのような定義をしたとしてもユニーク解の個数それ自体は変
 わりません。
 
   さて、Ｎクイーン問題は正方形のボードで形成されるので回転・反転による変換パター
 ンはぜんぶで８通りあります。だからといって「全解数＝ユニーク解数×８」と単純には
 いきません。ひとつのグループの要素数が必ず８個あるとは限らないのです。Ｎ＝５の
 下の例では要素数が２個のものと８個のものがあります。


 Ｎ＝５の全解は１０、ユニーク解は２なのです。
 
 グループ１: ユニーク解１つ目
 - - - Q -   - Q - - -
 Q - - - -   - - - - Q
 - - Q - -   - - Q - -
 - - - - Q   Q - - - -
 - Q - - -   - - - Q -
 
 グループ２: ユニーク解２つ目
 - - - - Q   Q - - - -   - - Q - -   - - Q - -   - - - Q -   - Q - - -   Q - - - -   - - - - Q
 - - Q - -   - - Q - -   Q - - - -   - - - - Q   - Q - - -   - - - Q -   - - - Q -   - Q - - -
 Q - - - -   - - - - Q   - - - Q -   - Q - - -   - - - - Q   Q - - - -   - Q - - -   - - - Q -
 - - - Q -   - Q - - -   - Q - - -   - - - Q -   - - Q - -   - - Q - -   - - - - Q   Q - - - -
 - Q - - -   - - - Q -   - - - - Q   Q - - - -   Q - - - -   - - - - Q   - - Q - -   - - Q - -

 
   それでは、ユニーク解を判定するための定義付けを行いますが、次のように定義する
 ことにします。各行のクイーンが右から何番目にあるかを調べて、最上段の行から下
 の行へ順番に列挙します。そしてそれをＮ桁の数値として見た場合に最小値になるもの
 をユニーク解として数えることにします。尚、このＮ桁の数を以後は「ユニーク判定値」
 と呼ぶことにします。
 
 - - - - Q   0
 - - Q - -   2
 Q - - - -   4   --->  0 2 4 1 3  (ユニーク判定値)
 - - - Q -   1
 - Q - - -   3
 
 
   探索によって得られたある１つの解(オリジナル)がユニーク解であるかどうかを判定
 するには「８通りの変換を試み、その中でオリジナルのユニーク判定値が最小であるか
 を調べる」ことになります。しかし結論から先にいえば、ユニーク解とは成り得ないこ
 とが明確なパターンを探索中に切り捨てるある枝刈りを組み込むことにより、３通りの
 変換を試みるだけでユニーク解の判定が可能になります。
  
 
 ■ユニーク解の個数を求める
   先ず最上段の行のクイーンの位置に着目します。その位置が左半分の領域にあればユ
 ニーク解には成り得ません。何故なら左右反転によって得られるパターンのユニーク判
 定値の方が確実に小さくなるからです。また、Ｎが奇数の場合に中央にあった場合はど
 うでしょう。これもユニーク解には成り得ません。何故なら仮に中央にあった場合、そ
 れがユニーク解であるためには少なくとも他の外側の３辺におけるクイーンの位置も中
 央になければならず、それは互いの効き筋にあたるので有り得ません。

  ***********************************************************************
  最上段の行のクイーンの位置は中央を除く右側の領域に限定されます。(ただし、N ≧ 2)
  ***********************************************************************
  
    次にその中でも一番右端(右上の角)にクイーンがある場合を考えてみます。他の３つ
  の角にクイーンを置くことはできないので(効き筋だから）、ユニーク解であるかどうか
  を判定するには、右上角から左下角を通る斜軸で反転させたパターンとの比較だけになり
  ます。突き詰めれば、
  
  [上から２行目のクイーンの位置が右から何番目にあるか]
  [右から２列目のクイーンの位置が上から何番目にあるか]
  
 
  を比較するだけで判定することができます。この２つの値が同じになることはないからです。
  
        3 0
        ↓↓
  - - - - Q ←0
  - Q - - - ←3
  - - - - -         上から２行目のクイーンの位置が右から４番目にある。
  - - - Q -         右から２列目のクイーンの位置が上から４番目にある。
  - - - - -         しかし、互いの効き筋にあたるのでこれは有り得ない。
  
    結局、再帰探索中において下図の X への配置を禁止する枝刈りを入れておけば、得
  られる解は総てユニーク解であることが保証されます。
  
  - - - - X Q
  - Q - - X -
  - - - - X -
  - - - - X -
  - - - - - -
  - - - - - -
  
    次に右端以外にクイーンがある場合を考えてみます。オリジナルがユニーク解である
  ためには先ず下図の X への配置は禁止されます。よって、その枝刈りを先ず入れておき
  ます。
  
  X X - - - Q X X
  X - - - - - - X
  - - - - - - - -
  - - - - - - - -
  - - - - - - - -
  - - - - - - - -
  X - - - - - - X
  X X - - - - X X
  
    次にクイーンの利き筋を辿っていくと、結局、オリジナルがユニーク解ではない可能
  性があるのは、下図の A,B,C の位置のどこかにクイーンがある場合に限られます。従っ
  て、90度回転、180度回転、270度回転の３通りの変換パターンだけを調べれはよいこと
  になります。
  
  X X x x x Q X X
  X - - - x x x X
  C - - x - x - x
  - - x - - x - -
  - x - - - x - -
  x - - - - x - A
  X - - - - x - X
  X X B - - x X X
 
 
  ■ユニーク解から全解への展開
    これまでの考察はユニーク解の個数を求めるためのものでした。全解数を求めるには
  ユニーク解を求めるための枝刈りを取り除いて全探索する必要があります。したがって
  探索時間を犠牲にしてしまうことになります。そこで「ユニーク解の個数から全解数を
  導いてしまおう」という試みが考えられます。これは、左右反転によるパターンの探索
  を省略して最後に結果を２倍するというアイデアの拡張版といえるものです。そしてそ
  れを実現させるには「あるユニーク解が属するグループの要素数はいくつあるのか」と
  いう考察が必要になってきます。
  
    最初に、クイーンが右上角にあるユニーク解を考えます。斜軸で反転したパターンが
  オリジナルと同型になることは有り得ないことと(×２)、右上角のクイーンを他の３つの
  角に写像させることができるので(×４)、このユニーク解が属するグループの要素数は必
  ず８個(＝２×４)になります。
  
    次に、クイーンが右上角以外にある場合は少し複雑になりますが、考察を簡潔にする
  ために次の事柄を確認します。
 
  TOTAL = (COUNT8 * 8) + (COUNT4 * 4) + (COUNT2 * 2);
    (1) 90度回転させてオリジナルと同型になる場合、さらに90度回転(オリジナルか
     ら180度回転)させても、さらに90度回転(オリジナルから270度回転)させてもオリ
     ジナルと同型になる。  
 
     COUNT2 * 2
  
    (2) 90度回転させてオリジナルと異なる場合は、270度回転させても必ずオリジナ
     ルとは異なる。ただし、180度回転させた場合はオリジナルと同型になることも有
     り得る。 
 
     COUNT4 * 4
  
    (3) (1) に該当するユニーク解が属するグループの要素数は、左右反転させたパターンを
        加えて２個しかありません。(2)に該当するユニーク解が属するグループの要素数は、
        180度回転させて同型になる場合は４個(左右反転×縦横回転)、そして180度回転させても
        オリジナルと異なる場合は８個になります。(左右反転×縦横回転×上下反転)
  
     COUNT8 * 8 
 
    以上のことから、ひとつひとつのユニーク解が上のどの種類に該当するのかを調べる
  ことにより全解数を計算で導き出すことができます。探索時間を短縮させてくれる枝刈
  りを外す必要がなくなったというわけです。 
  
    UNIQUE  COUNT2      +  COUNT4      +  COUNT8
    TOTAL  (COUNT2 * 2) + (COUNT4 * 4) + (COUNT8 * 8)
 
  　これらを実現すると、前回のNQueen3()よりも実行速度が遅くなります。
  　なぜなら、対称・反転・斜軸を反転するための処理が加わっているからです。
  ですが、今回の処理を行うことによって、さらにNQueen5()では、処理スピードが飛
  躍的に高速化されます。そのためにも今回のアルゴリズム実装は必要なのです。




実行結果

$ nvcc -O3 CUDA04_N-Queen.cu  && ./a.out -r
４．CPUR 再帰 バックトラック＋対称解除法
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.01
12:        14200            1787            0.05
13:        73712            9233            0.28
14:       365596           45752            1.65
15:      2279184          285053           10.61
16:     14772512         1846955         1:12.29
17:     95815104        11977939         8:42.16

$ nvcc -O3 CUDA04_N-Queen.cu  && ./a.out -c
４．CPU 非再帰 バックトラック＋対称解除法
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.01
12:        14200            1787            0.05
13:        73712            9233            0.28
14:       365596           45752            1.71
15:      2279184          285053           10.92
16:     14772512         1846955         1:13.64
17:     95815104        11977939         8:46.42

$ nvcc -O3 CUDA04_N-Queen.cu  && ./a.out -g
４．GPU 非再帰 バックトラック＋対称解除法
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               1  00:00:00:00.02
 5:           10               2  00:00:00:00.00
 6:            4               1  00:00:00:00.00
 7:           40               6  00:00:00:00.00
 8:           92              12  00:00:00:00.01
 9:          352              46  00:00:00:00.07
10:          724              92  00:00:00:00.29
11:         2680             341  00:00:00:01.13
12:        14200            1787  00:00:00:05.29
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
//変数宣言
int aBoard[MAX];
int down[2*MAX-1];  //down:flagA 縦 配置フラグ　
int left[2*MAX-1];  //left:flagB 斜め配置フラグ　
int right[2*MAX-1]; //right:flagC 斜め配置フラグ　
long TOTAL=0;       //CPU,CPUR
long UNIQUE=0;      //CPU,CPUR      
int aT[MAX];       //aT:aTrial[]
int aS[MAX];       //aS:aScrath[]
//関数宣言 SGPU
__global__ void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond);
long long sgpu_solve_nqueen_cuda(int size,int steps); 
//関数宣言 GPU
__global__
void nqueen_cuda(int *d_aBoard,int *d_aT,int *d_aS,int *d_down,int *d_right,int *d_left,long *d_results,long TOTAL,long UNIQUE,int row,int size);
void solve_nqueen_cuda(int si,long results[2],int steps);
__device__
int symmetryOps(int size,int *d_aBoard,int *d_aT,int *d_aS);
//関数宣言 GPU/CPU
__device__ __host__
void rotate(int chk[],int scr[],int n,int neg);
__device__ __host__
void vMirror(int chk[],int n);
__device__ __host__
int intncmp(int lt[],int rt[],int n);
//関数宣言 CPU
void TimeFormat(clock_t utime,char *form);
int symmetryOps(int si);
void NQueen(int row,int size);
void NQueenR(int row,int size);
//
__global__ void sgpu_cuda_kernel(
    int size,int mark,
    unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,
    unsigned int* results,int totalCond){
  const int tid=threadIdx.x;
  const int bid=blockIdx.x;
  const int idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10];
  __shared__ unsigned int left[THREAD_NUM][10];
  __shared__ unsigned int right[THREAD_NUM][10];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;
  unsigned int bit;
  if(idx<totalCond){
    down[tid][row]=totalDown[idx];
    left[tid][row]=totalLeft[idx];
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];
    while(row>=0){
      if((bitmap[tid][row]&mask)==mask){row--;}
      else{
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        if((bit&mask)!=0){
          if(row+1==mark){total++;row--;}
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{row--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
//
long long sgpu_solve_nqueen_cuda(int size,int steps) {
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;
  unsigned int* leftCuda;
  unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}	
  total*=2;


  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
      >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] results;
  return total;
}
//
//対称解除法
__device__
int symmetryOps(int size,int *d_aBoard,int *d_aT,int *d_aS){
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    d_aT[i]=d_aBoard[i];
  }
  //時計回りに90度回転
  rotate(d_aT,d_aS,size,0);       
  int k=intncmp(d_aBoard,d_aT,size);
  if(k>0) return 0;
  if(k==0){
    nEquiv=1;
  }else{
    //時計回りに180度回転
    rotate(d_aT,d_aS,size,0);     
    k=intncmp(d_aBoard,d_aT,size);
    if(k>0) return 0;
    if(k==0){
      nEquiv=2;
    }else{
      //時計回りに270度回転
      rotate(d_aT,d_aS,size,0);   
      k=intncmp(d_aBoard,d_aT,size);
      if(k>0){
        return 0;
      }
      nEquiv=4;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    d_aT[i]=d_aBoard[i];
  }
  //垂直反転
  vMirror(d_aT,size);           
  k=intncmp(d_aBoard,d_aT,size);
  if(k>0){
    return 0;
  }
  //-90度回転 対角鏡と同等
  if(nEquiv>1){             
    rotate(d_aT,d_aS,size,1);
    k=intncmp(d_aBoard,d_aT,size);
    if(k>0){
      return 0;
    }
    //-180度回転 水平鏡像と同等
    if(nEquiv>2){           
      rotate(d_aT,d_aS,size,1);
      k=intncmp(d_aBoard,d_aT,size);
      if(k>0){
        return 0;
      }  
      //-270度回転 反対角鏡と同等
      rotate(d_aT,d_aS,size,1);
      k=intncmp(d_aBoard,d_aT,size);
      if(k>0){
        return 0;
      }
    }
  }
  return nEquiv*2;
}
//
__global__
void nqueen_cuda(int *d_aBoard,int *d_aT,int *d_aS,int *d_down,int *d_right,int *d_left,long *d_results,long TOTAL,long UNIQUE,int row,int size){
   int sizeE=size-1;
  bool matched;
  while(row>=0){
    matched=false;
    // １回目はaBoard[row]が-1なのでcolを0で初期化
    // ２回目以降はcolを<sizeまで右へシフト
    for(int col=d_aBoard[row]+1;col<size;col++){
      if(d_down[col]==0
          && d_right[col-row+sizeE]==0
          && d_left[col+row]==0){ //まだ効き筋がない
        if(d_aBoard[row]!=-1){    //Qを配置済み
          //colがaBoard[row]におきかわる
          d_down[d_aBoard[row]]
            =d_right[d_aBoard[row]-row+sizeE]
            =d_left[d_aBoard[row]+row]=0;
        }
        d_aBoard[row]=col;        //Qを配置
        d_down[col]
          =d_right[col-row+sizeE]
          =d_left[col+row]=1;     //効き筋とする
        matched=true;           //配置した
        break;
      }
    }
    if(matched){                //配置済みなら
      row++;                    //次のrowへ
      if(row==size){
        //print(size); //print()でTOTALを++しない
        /** 対称解除法の導入 */
        int s=symmetryOps(size,d_aBoard,d_aT,d_aS);
        if(s!=0){
          //print(size); //print()でTOTALを++しない
          //ホストに戻す配列にTOTALを入れる
          //スレッドが１つの場合は配列は１個
          d_results[1]=++UNIQUE; 
          d_results[0]+=s;   //対称解除で得られた解数を加算
        }
        /** 対称解除法の導入 */
        row--;
      }
    }else{
      if(d_aBoard[row]!=-1){
        int col=d_aBoard[row];    /** col の代用 */
        d_down[col]
          =d_right[col-row+sizeE]
          =d_left[col+row]=0;
        d_aBoard[row]=-1;
      }
      row--;                    //バックトラック
    }
  }
}
//
void solve_nqueen_cuda(int si,long results[2],int steps){
  //メモリ登録
  int *h_aBoard;
  int *h_aT;
  int *h_aS;
  int *h_down;
  int *h_right;
  int *h_left;
  long *h_results;
  hipHostMalloc((void**)&h_aBoard,sizeof(int)*MAX, hipHostMallocDefault);
  hipHostMalloc((void**)&h_aT,sizeof(int)*MAX, hipHostMallocDefault);
  hipHostMalloc((void**)&h_aS,sizeof(int)*MAX, hipHostMallocDefault);
  hipHostMalloc((void**)&h_down,sizeof(int)*2*MAX-1, hipHostMallocDefault);
  hipHostMalloc((void**)&h_right,sizeof(int)*2*MAX-1, hipHostMallocDefault);
  hipHostMalloc((void**)&h_left,sizeof(int)*2*MAX-1, hipHostMallocDefault);
  hipHostMalloc((void**)&h_results,sizeof(long)*steps, hipHostMallocDefault);
  int *d_aBoard;
  int *d_aT;
  int *d_aS;
  int *d_down;
  int *d_right;
  int *d_left;
  long *d_results;
  hipMalloc((void**)&d_aBoard,sizeof(int)*MAX);
  hipMalloc((void**)&d_aT,sizeof(int)*MAX);
  hipMalloc((void**)&d_aS,sizeof(int)*MAX);
  hipMalloc((void**)&d_down,sizeof(int)*2*MAX-1);
  hipMalloc((void**)&d_right,sizeof(int)*2*MAX-1);
  hipMalloc((void**)&d_left,sizeof(int)*2*MAX-1);
  hipMalloc((void**)&d_results,sizeof(long)*steps);
  //初期化
  for(int i=0;i<si;i++){
      h_aBoard[i]=-1;
  }
  //host to device
  hipMemcpy(d_aBoard,h_aBoard,
    sizeof(int)*MAX,hipMemcpyHostToDevice);
  hipMemcpy(d_aT,h_aT,
    sizeof(int)*MAX,hipMemcpyHostToDevice);
  hipMemcpy(d_aS,h_aS,
    sizeof(int)*MAX,hipMemcpyHostToDevice);
  hipMemcpy(d_down,h_down,
    sizeof(int)*2*MAX-1,hipMemcpyHostToDevice);
  hipMemcpy(d_right,h_right,
    sizeof(int)*2*MAX-1,hipMemcpyHostToDevice);
  hipMemcpy(d_left,h_left,
    sizeof(int)*2*MAX-1,hipMemcpyHostToDevice);
  hipMemcpy(d_results,h_results,
    sizeof(long)*steps,hipMemcpyHostToDevice);
  //実行
  nqueen_cuda<<<1,1>>>(d_aBoard,d_aT,d_aS,d_down,d_right,d_left,d_results,0,0,0,si);
  //device to host
  hipMemcpy(h_results,d_results,
    sizeof(long)*steps,hipMemcpyDeviceToHost);
  //結果の格納
  results[0]=h_results[0];
  results[1]=h_results[1];
  //メモリの開放
  hipHostFree(h_aBoard);
  hipHostFree(h_aT);
  hipHostFree(h_aS);
  hipHostFree(h_down);
  hipHostFree(h_right);
  hipHostFree(h_left);
  hipHostFree(h_results);
  hipFree(d_aBoard);
  hipFree(d_aT);
  hipFree(d_aS);
  hipFree(d_down);
  hipFree(d_left);
  hipFree(d_right);
  hipFree(d_results);
}
/** CUDA 初期化 **/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form){
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//回転
__device__ __host__
void rotate(int chk[],int scr[],int n,int neg){
  int k=neg ? 0 : n-1;
  int incr=(neg ? +1 : -1);
  for(int j=0;j<n;k+=incr){
    scr[j++]=chk[k];
  }
  k=neg ? n-1 : 0;
  for(int j=0;j<n;k-=incr){
    chk[scr[j++]]=k;
  }
}
//反転
__device__ __host__
void vMirror(int chk[],int n){
  for(int j=0;j<n;j++){
    chk[j]=(n-1)-chk[j];
  }
}
//
__device__ __host__
int intncmp(int lt[],int rt[],int n){
  int rtn=0;
  for(int k=0;k<n;k++){
    rtn=lt[k]-rt[k];
    if(rtn!=0){
      break;
    }
  }
  return rtn;
}
//対称解除法
int symmetryOps(int size){
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    aT[i]=aBoard[i];
  }
  //時計回りに90度回転
  rotate(aT,aS,size,0);       
  int k=intncmp(aBoard,aT,size);
  if(k>0) return 0;
  if(k==0){
    nEquiv=1;
  }else{
    //時計回りに180度回転
    rotate(aT,aS,size,0);     
    k=intncmp(aBoard,aT,size);
    if(k>0) return 0;
    if(k==0){
      nEquiv=2;
    }else{
      //時計回りに270度回転
      rotate(aT,aS,size,0);   
      k=intncmp(aBoard,aT,size);
      if(k>0){
        return 0;
      }
      nEquiv=4;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    aT[i]=aBoard[i];
  }
  //垂直反転
  vMirror(aT,size);           
  k=intncmp(aBoard,aT,size);
  if(k>0){
    return 0;
  }
  //-90度回転 対角鏡と同等
  if(nEquiv>1){             
    rotate(aT,aS,size,1);
    k=intncmp(aBoard,aT,size);
    if(k>0){
      return 0;
    }
    //-180度回転 水平鏡像と同等
    if(nEquiv>2){           
      rotate(aT,aS,size,1);
      k=intncmp(aBoard,aT,size);
      if(k>0){
        return 0;
      }  
      //-270度回転 反対角鏡と同等
      rotate(aT,aS,size,1);
      k=intncmp(aBoard,aT,size);
      if(k>0){
        return 0;
      }
    }
  }
  return nEquiv*2;
}
// CPU 非再帰版 ロジックメソッド
void NQueen(int row,int size){
  int sizeE=size-1;
  bool matched;
  while(row>=0){
    matched=false;
    // １回目はaBoard[row]が-1なのでcolを0で初期化
    // ２回目以降はcolを<sizeまで右へシフト
    for(int col=aBoard[row]+1;col<size;col++){
      if(down[col]==0
          && right[col-row+sizeE]==0
          && left[col+row]==0){ //まだ効き筋がない
        if(aBoard[row]!=-1){    //Qを配置済み
          //colがaBoard[row]におきかわる
          down[aBoard[row]]
            =right[aBoard[row]-row+sizeE]
            =left[aBoard[row]+row]=0;
        }
        aBoard[row]=col;        //Qを配置
        down[col]
          =right[col-row+sizeE]
          =left[col+row]=1;     //効き筋とする
        matched=true;           //配置した
        break;
      }
    }
    if(matched){                //配置済みなら
      row++;                    //次のrowへ
      if(row==size){
        //print(size); //print()でTOTALを++しない
        /** 対称解除法の導入 */
        int s=symmetryOps(size);
        if(s!=0){
          UNIQUE++;             //ユニーク解を加算
          TOTAL+=s;   //対称解除で得られた解数を加算
        }
        // TOTAL++;
        /** 対称解除法の導入 */
        row--;
      }
    }else{
      if(aBoard[row]!=-1){
        int col=aBoard[row];    /** col の代用 */
        down[col]
          =right[col-row+sizeE]
          =left[col+row]=0;
        aBoard[row]=-1;
      }
      row--;                    //バックトラック
    }
  }
}
// CPUR 再帰版 ロジックメソッド
void NQueenR(int row,int size){
  int sizeE=size-1;
  if(row==size){
    /** 対称解除法の導入 */
    int s=symmetryOps(size);
    if(s!=0){
      UNIQUE++;       //ユニーク解を加算
      TOTAL+=s;       //対称解除で得られた解数を加算
    }
    // TOTAL++;
    /** 対称解除法の導入 */
  }else{
    for(int col=aBoard[row]+1;col<size;col++){
      aBoard[row]=col;
      if(down[col]==0
          && right[row-col+sizeE]==0
          && left[row+col]==0){
        down[col]
          =right[row-col+sizeE]
          =left[row+col]=1;
        NQueenR(row+1,size);
        down[col]
          =right[row-col+sizeE]
          =left[row+col]=0;
      }
      aBoard[row]=-1;
    }
  }
}
//メインメソッド
int main(int argc,char** argv) {
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  if(cpu){
    printf("\n\n４．CPU 非再帰 バックトラック＋対称解除法\n");
  }else if(cpur){
    printf("\n\n４．CPUR 再帰 バックトラック＋対称解除法\n");
  }else if(gpu){
    printf("\n\n４．GPU 非再帰 バックトラック＋対称解除法\n");
  }else if(sgpu){
    printf("\n\n４．SGPU 非再帰 バックトラック＋対称解除法\n");
  }
  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;           //速度計測用
    char t[20];           //hh:mm:ss.msを格納
    int min=4; 
    int targetN=18;
    for(int i=min;i<=targetN;i++){
      for(int j=0;j<=targetN;j++){ aBoard[j]=-1; }
      TOTAL=0; 
      UNIQUE=0;
      st=clock();
      if(cpu){ NQueen(0,i); }
      if(cpur){ NQueenR(0,i); }
      TimeFormat(clock()-st,t); 
      printf("%2d:%13ld%16ld%s\n",i,TOTAL,UNIQUE,t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=18;
    struct timeval t0;
    struct timeval t1;
    int ss;int ms;int dd;
    long TOTAL,UNIQUE;
    long results[2];//結果格納用
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        solve_nqueen_cuda(i,results,steps);
        TOTAL=results[0];
        UNIQUE=results[1];
      }else if(sgpu){
        TOTAL=sgpu_solve_nqueen_cuda(i,steps);
      }
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
