/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc -O3 CUDA**_N-Queen.cu && ./a.out (-c|-r|-g|-s)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 
                    -s SGPU(サマーズ版と思われる)

 * ５．バックトラック＋対称解除法＋枝刈りと最適化

 * 　単純ですのでソースのコメントを見比べて下さい。
 *   単純ではありますが、枝刈りの効果は絶大です。

実行結果
$ nvcc -O3 CUDA05_N-Queen.cu  && ./a.out -r
５．CPUR 再帰 バックトラック＋対称解除法＋枝刈りと最適化
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.01
12:        14200            1787            0.03
13:        73712            9233            0.16
14:       365596           45752            0.85
15:      2279184          285053            5.88
16:     14772512         1846955           37.43
17:     95815104        11977939         5:52.47

$ nvcc -O3 CUDA05_N-Queen.cu  && ./a.out -c
５．CPU 非再帰 バックトラック＋対称解除法＋枝刈りと最適化
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.01
12:        14200            1787            0.03
13:        73712            9233            0.16
14:       365596           45752            0.89
15:      2279184          285053            6.55
16:     14772512         1846955           50.88
17:     95815104        11977939         6:12.60

$ nvcc -O3 CUDA05_N-Queen.cu  && ./a.out -g
５．GPU 再帰 バックトラック＋対称解除法＋枝刈りと最適化
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               1  00:00:00:00.02
 5:           10               2  00:00:00:00.00
 6:            4               1  00:00:00:00.00
 7:           40               6  00:00:00:00.00
 8:           92              12  00:00:00:00.01
 9:          352              46  00:00:00:00.04
10:          724              92  00:00:00:00.15
11:         2680             341  00:00:00:00.70
12:        14200            1787  00:00:00:02.71
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


//
#define THREAD_NUM		96
#define MAX 27
//変数宣言
int aBoard[MAX];
int down[2*MAX-1];  //down:flagA 縦 配置フラグ　
int left[2*MAX-1];  //left:flagB 斜め配置フラグ　
int right[2*MAX-1]; //right:flagC 斜め配置フラグ　
long TOTAL=0;       //CPU,CPUR
long UNIQUE=0;      //CPU,CPUR
int aT[MAX];        //aT:aTrial[]
int aS[MAX];        //aS:aScrath[]
//関数宣言 GPU
__global__
void nqueen_cuda(int *d_aBoard,int *d_aT,int *d_aS,int *d_down,int *d_right,int *d_left,long *d_results,long TOTAL,long UNIQUE,int row,int size);
void solve_nqueen_cuda(int si,long results[2],int steps);
__device__
int symmetryOps(int size,int *d_aBoard,int *d_aT,int *d_aS);
//関数宣言 SGPU
__global__ void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond);
long long sgpu_solve_nqueen_cuda(int size,int steps); 
//関数宣言 GPU/CPU
__device__ __host__
void rotate(int chk[],int scr[],int n,int neg);
__device__ __host__
void vMirror(int chk[],int n);
__device__ __host__
int intncmp(int lt[],int rt[],int n);
//関数宣言 CPU
void TimeFormat(clock_t utime,char *form);
int symmetryOps(int si);
void NQueen(int row,int size);
void NQueenR(int row,int size);
//
__global__ void sgpu_cuda_kernel(
    int size,int mark,
    unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,
    unsigned int* results,int totalCond){
  const int tid=threadIdx.x;
  const int bid=blockIdx.x;
  const int idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10];
  __shared__ unsigned int left[THREAD_NUM][10];
  __shared__ unsigned int right[THREAD_NUM][10];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;
  unsigned int bit;
  if(idx<totalCond){
    down[tid][row]=totalDown[idx];
    left[tid][row]=totalLeft[idx];
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];
    while(row>=0){
      if((bitmap[tid][row]&mask)==mask){row--;}
      else{
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        if((bit&mask)!=0){
          if(row+1==mark){total++;row--;}
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{row--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
//
long long sgpu_solve_nqueen_cuda(int size,int steps) {
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;
  unsigned int* leftCuda;
  unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}	
  total*=2;


  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
      >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] results;
  return total;
}

/** CUDA 初期化 **/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form){
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//回転
__device__ __host__
void rotate(int chk[],int scr[],int n,int neg){
  int k=neg ? 0 : n-1;
  int incr=(neg ? +1 : -1);
  for(int j=0;j<n;k+=incr){
    scr[j++]=chk[k];
  }
  k=neg ? n-1 : 0;
  for(int j=0;j<n;k-=incr){
    chk[scr[j++]]=k;
  }
}
//反転
__device__ __host__
void vMirror(int chk[],int n){
  for(int j=0;j<n;j++){
    chk[j]=(n-1)-chk[j];
  }
}
//
__device__ __host__
int intncmp(int lt[],int rt[],int n){
  int rtn=0;
  for(int k=0;k<n;k++){
    rtn=lt[k]-rt[k];
    if(rtn!=0){
      break;
    }
  }
  return rtn;
}
//対称解除法
int symmetryOps(int size){
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    aT[i]=aBoard[i];
  }
  //時計回りに90度回転
  rotate(aT,aS,size,0);       
  int k=intncmp(aBoard,aT,size);
  if(k>0) return 0;
  if(k==0){
    nEquiv=1;
  }else{
    //時計回りに180度回転
    rotate(aT,aS,size,0);     
    k=intncmp(aBoard,aT,size);
    if(k>0) return 0;
    if(k==0){
      nEquiv=2;
    }else{
      //時計回りに270度回転
      rotate(aT,aS,size,0);   
      k=intncmp(aBoard,aT,size);
      if(k>0){
        return 0;
      }
      nEquiv=4;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    aT[i]=aBoard[i];
  }
  //垂直反転
  vMirror(aT,size);           
  k=intncmp(aBoard,aT,size);
  if(k>0){
    return 0;
  }
  //-90度回転 対角鏡と同等
  if(nEquiv>1){             
    rotate(aT,aS,size,1);
    k=intncmp(aBoard,aT,size);
    if(k>0){
      return 0;
    }
    //-180度回転 水平鏡像と同等
    if(nEquiv>2){           
      rotate(aT,aS,size,1);
      k=intncmp(aBoard,aT,size);
      if(k>0){
        return 0;
      }  //-270度回転 反対角鏡と同等
      rotate(aT,aS,size,1);
      k=intncmp(aBoard,aT,size);
      if(k>0){
        return 0;
      }
    }
  }
  return nEquiv*2;
}
__device__
int symmetryOps(int size,int *d_aBoard,int *d_aT,int *d_aS){
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    d_aT[i]=d_aBoard[i];
  }
  //時計回りに90度回転
  rotate(d_aT,d_aS,size,0);       
  int k=intncmp(d_aBoard,d_aT,size);
  if(k>0) return 0;
  if(k==0){
    nEquiv=1;
  }else{
    //時計回りに180度回転
    rotate(d_aT,d_aS,size,0);     
    k=intncmp(d_aBoard,d_aT,size);
    if(k>0) return 0;
    if(k==0){
      nEquiv=2;
    }else{
      //時計回りに270度回転
      rotate(d_aT,d_aS,size,0);   
      k=intncmp(d_aBoard,d_aT,size);
      if(k>0){
        return 0;
      }
      nEquiv=4;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    d_aT[i]=d_aBoard[i];
  }
  //垂直反転
  vMirror(d_aT,size);           
  k=intncmp(d_aBoard,d_aT,size);
  if(k>0){
    return 0;
  }
  //-90度回転 対角鏡と同等
  if(nEquiv>1){             
    rotate(d_aT,d_aS,size,1);
    k=intncmp(d_aBoard,d_aT,size);
    if(k>0){
      return 0;
    }
    //-180度回転 水平鏡像と同等
    if(nEquiv>2){           
      rotate(d_aT,d_aS,size,1);
      k=intncmp(d_aBoard,d_aT,size);
      if(k>0){
        return 0;
      }  
      //-270度回転 反対角鏡と同等
      rotate(d_aT,d_aS,size,1);
      k=intncmp(d_aBoard,d_aT,size);
      if(k>0){
        return 0;
      }
    }
  }
  return nEquiv*2;
}
//
__global__
void nqueen_cuda(int *d_aBoard,int *d_aT,int *d_aS,int *d_down,int *d_right,int *d_left,long *d_results,long TOTAL,long UNIQUE,int row,int size){
  bool matched;
  int sizeE=size-1;
  while(row>=0){
    matched=false;
    /** 枝刈り */
    int lim=(row!=0)?size:(size+1)/2;
    for(int col=d_aBoard[row]+1;col<lim;col++){
      if(d_down[col]==0
          && d_right[col-row+sizeE]==0
          && d_left[col+row]==0){
        if(d_aBoard[row]!=-1){
          d_down[d_aBoard[row]]
            =d_right[d_aBoard[row]-row+sizeE]
            =d_left[d_aBoard[row]+row]=0;
        }
        d_aBoard[row]=col;
        d_down[col]
          =d_right[col-row+sizeE]
          =d_left[col+row]=1;
        matched=true;
        break;
      }
    }
    if(matched){
      row++;
      if(row==size){
        int s=symmetryOps(size,d_aBoard,d_aT,d_aS);
        if(s!=0){
          //print(size); //print()でTOTALを++しない
          //ホストに戻す配列にTOTALを入れる
          //スレッドが１つの場合は配列は１個
          d_results[1]=++UNIQUE; 
          d_results[0]+=s;   //対称解除で得られた解数を加算
        }
        row--;
      }
    }else{
      if(d_aBoard[row]!=-1){
        int col=d_aBoard[row];
        d_down[col]
          =d_right[col-row+sizeE]
          =d_left[col+row]=0;
        d_aBoard[row]=-1;
      }
      row--;
    }
  }
}
//
void solve_nqueen_cuda(int si,long results[2],int steps){
  //メモリ管理 
  int *h_aBoard;
  int *h_aT;
  int *h_aS;
  int *h_down;
  int *h_right;
  int *h_left;
  long *h_results;
  hipHostMalloc((void**)&h_aBoard,sizeof(int)*MAX, hipHostMallocDefault);
  hipHostMalloc((void**)&h_aT,sizeof(int)*MAX, hipHostMallocDefault);
  hipHostMalloc((void**)&h_aS,sizeof(int)*MAX, hipHostMallocDefault);
  hipHostMalloc((void**)&h_down,sizeof(int)*2*MAX-1, hipHostMallocDefault);
  hipHostMalloc((void**)&h_right,sizeof(int)*2*MAX-1, hipHostMallocDefault);
  hipHostMalloc((void**)&h_left,sizeof(int)*2*MAX-1, hipHostMallocDefault);
  hipHostMalloc((void**)&h_results,sizeof(long)*steps, hipHostMallocDefault);
  int *d_aBoard;
  int *d_aT;
  int *d_aS;
  int *d_down;
  int *d_right;
  int *d_left;
  long *d_results;
  hipMalloc((void**)&d_aBoard,sizeof(int)*MAX);
  hipMalloc((void**)&d_aT,sizeof(int)*MAX);
  hipMalloc((void**)&d_aS,sizeof(int)*MAX);
  hipMalloc((void**)&d_down,sizeof(int)*2*MAX-1);
  hipMalloc((void**)&d_right,sizeof(int)*2*MAX-1);
  hipMalloc((void**)&d_left,sizeof(int)*2*MAX-1);
  hipMalloc((void**)&d_results,sizeof(long)*steps);
  //初期化
  for(int i=0;i<si;i++){
      h_aBoard[i]=-1;
  }
  //host to device
  hipMemcpy(d_aBoard,h_aBoard,
    sizeof(int)*MAX,hipMemcpyHostToDevice);
  hipMemcpy(d_aT,h_aT,
    sizeof(int)*MAX,hipMemcpyHostToDevice);
  hipMemcpy(d_aS,h_aS,
    sizeof(int)*MAX,hipMemcpyHostToDevice);
  hipMemcpy(d_down,h_down,
    sizeof(int)*2*MAX-1,hipMemcpyHostToDevice);
  hipMemcpy(d_right,h_right,
    sizeof(int)*2*MAX-1,hipMemcpyHostToDevice);
  hipMemcpy(d_left,h_left,
    sizeof(int)*2*MAX-1,hipMemcpyHostToDevice);
  hipMemcpy(d_results,h_results,
    sizeof(long)*steps,hipMemcpyHostToDevice);
  //実行
  nqueen_cuda<<<1,1>>>(d_aBoard,d_aT,d_aS,d_down,d_right,d_left,d_results,0,0,0,si);
  //device to host
  hipMemcpy(h_results,d_results,
    sizeof(long)*steps,hipMemcpyDeviceToHost);
  //結果の代入
  results[0]=h_results[0];
  results[1]=h_results[1];
  //メモリ解放
  hipHostFree(h_aBoard);
  hipHostFree(h_aT);
  hipHostFree(h_aS);
  hipHostFree(h_down);
  hipHostFree(h_right);
  hipHostFree(h_left);
  hipHostFree(h_results);
  hipFree(d_aBoard);
  hipFree(d_aT);
  hipFree(d_aS);
  hipFree(d_down);
  hipFree(d_left);
  hipFree(d_right);
  hipFree(d_results);
}
//
//CPU 非再帰版 ロジックメソッド
void NQueen(int row,int size){
  bool matched;
  int sizeE=size-1;
  while(row>=0){
    matched=false;
    /** 枝刈り */
    int lim=(row!=0)?size:(size+1)/2;
    for(int col=aBoard[row]+1;col<lim;col++){
      if(down[col]==0
          && right[col-row+sizeE]==0
          && left[col+row]==0){
        if(aBoard[row]!=-1){
          down[aBoard[row]]
            =right[aBoard[row]-row+sizeE]
            =left[aBoard[row]+row]=0;
        }
        aBoard[row]=col;
        down[col]
          =right[col-row+sizeE]
          =left[col+row]=1;
        matched=true;
        break;
      }
    }
    if(matched){
      row++;
      if(row==size){
        int s=symmetryOps(size);
        if(s!=0){
          UNIQUE++;
          TOTAL+=s;
        }
        row--;
      }
    }else{
      if(aBoard[row]!=-1){
        int col=aBoard[row];
        down[col]
          =right[col-row+sizeE]
          =left[col+row]=0;
        aBoard[row]=-1;
      }
      row--;
    }
  }
}
//CPUR 再帰版 ロジックメソッド
void NQueenR(int row,int size){
  int sizeE=size-1;
  if(row==size){
    int s=symmetryOps(size);  //対称解除法の導入
    if(s!=0){
      UNIQUE++;
      TOTAL+=s;
    }
  }else{
    /** 枝刈り */
    int lim=(row!=0) ? size : (size+1)/2;
    for(int col=aBoard[row]+1;col<lim;col++){
      aBoard[row]=col;
      if(down[col]==0
          && right[row-col+sizeE]==0
          && left[row+col]==0){
        down[col]
          =right[row-col+sizeE]
          =left[row+col]=1;
        NQueenR(row+1,size);
        down[col]
          =right[row-col+sizeE]
          =left[row+col]=0;
      }
      aBoard[row]=-1;
    }
  }
}
//メインメソッド
int main(int argc,char** argv) {
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  if(cpu){
    printf("\n\n５．CPU 非再帰 バックトラック＋対称解除法＋枝刈りと最適化\n");
  }else if(cpur){
    printf("\n\n５．CPUR 再帰 バックトラック＋対称解除法＋枝刈りと最適化\n");
  }else if(gpu){
    printf("\n\n５．GPU 非再帰 バックトラック＋対象解除法＋枝刈りと最適化\n");
  }else if(sgpu){
    printf("\n\n５．SGPU 非再帰 バックトラック＋対象解除法＋枝刈りと最適化\n");
  }
  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;           //速度計測用
    char t[20];           //hh:mm:ss.msを格納
    int min=4; int 
      targetN=18;
    for(int i=min;i<=targetN;i++){
      //aBoard配列を-1で初期化
      for(int j=0;j<=targetN;j++){ aBoard[j]=-1; }
      TOTAL=0; 
      UNIQUE=0;
      st=clock();
      if(cpu){ NQueen(0,i); }
      if(cpur){ NQueenR(0,i); }
      TimeFormat(clock()-st,t); 
      printf("%2d:%13ld%16ld%s\n",i,TOTAL,UNIQUE,t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=18;
    struct timeval t0;
    struct timeval t1;
    int ss;int ms;int dd;
    long TOTAL,UNIQUE;
    long results[2];//結果格納用
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        solve_nqueen_cuda(i,results,steps);
        TOTAL=results[0];
        UNIQUE=results[1];
      }else if(sgpu){
        TOTAL=sgpu_solve_nqueen_cuda(i,steps);
      }
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
