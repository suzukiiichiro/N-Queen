/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc -O3 CUDA**_N-Queen.cu && ./a.out (-c|-r|-g|-s)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 
                    -s SGPU(サマーズ版と思われる)

 １３．並列処理 pthread


 【注意】

 こちらのソースは参考版です

 理由：
 cudaのソースに参考のために 再帰・非再帰のpthread版を記載しましたが、
 cudaはpthreadをサポートしていないので、コンパイルは通りません。

 +590 行目のコメントアウトはそのためです。
 //iFbRet = pthread_create(&pth, NULL,&NQueenThread,NULL);

 pthreadの具体的なソースは、Cディレクトリの C13_N-Queen.cを見て下さい。
 こちらはきちんと動作します。
 【注意】



 実行結果

１３．CPUR 再帰 並列処理 pthread
 N:           Total           Unique          dd:hh:mm:ss.ms
 4:               2                1          00:00:00:00.00
 5:              10                2          00:00:00:00.00
 6:               4                1          00:00:00:00.00
 7:              40                6          00:00:00:00.00
 8:              92               12          00:00:00:00.00
 9:             352               46          00:00:00:00.00
10:             724               92          00:00:00:00.00
11:            2680              341          00:00:00:00.00
12:           14200             1787          00:00:00:00.00
13:           73712             9233          00:00:00:00.00
14:          365596            45752          00:00:00:00.02
15:         2279184           285053          00:00:00:00.10
16:        14772512          1846955          00:00:00:00.63
17:        95815104         11977939          00:00:00:04.33

１３．CPU 非再帰 並列処理 pthread
 N:           Total           Unique          dd:hh:mm:ss.ms
 4:               2                1          00:00:00:00.00
 5:              10                2          00:00:00:00.00
 6:               4                1          00:00:00:00.00
 7:              40                6          00:00:00:00.00
 8:              92               12          00:00:00:00.00
 9:             352               46          00:00:00:00.00
10:             724               92          00:00:00:00.00
11:            2680              341          00:00:00:00.00
12:           14200             1787          00:00:00:00.00
13:           73712             9233          00:00:00:00.00
14:          365596            45752          00:00:00:00.01
15:         2279184           285053          00:00:00:00.10
16:        14772512          1846955          00:00:00:00.65
17:        95815104         11977939          00:00:00:04.33


１３．GPU 非再帰 並列処理 CUDA
＜＜うごきません＞＞

ちなみに１２の処理結果

bash-3.2$ nvcc -O3 CUDA12_N-Queen.cu && ./a.out -g
１２．GPU 非再帰 枝刈り
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               1  00:00:00:00.02
 5:           10               2  00:00:00:00.00
 6:            4               1  00:00:00:00.00
 7:           40               6  00:00:00:00.01
 8:           92              12  00:00:00:00.00
 9:          352              46  00:00:00:00.01
10:          724              92  00:00:00:00.01
11:         2680             341  00:00:00:00.02
12:        14200            1787  00:00:00:00.04
13:        73712            9233  00:00:00:00.06
14:       365596           45752  00:00:00:00.11
15:      2279184          285053  00:00:00:00.37
16:     14772512         1846955  00:00:00:01.77
17:     95815104        11977939  00:00:00:11.91
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#include <pthread.h>
//
#define THREAD_NUM		96
#define MAX 27
//

int NR;
// GPUで使います
long Total=0 ;      //合計解
long Unique=0;

//
//変数宣言
// pthreadはパラメータを１つしか渡せないので構造体に格納
//グローバル構造体
typedef struct {
  int size;
  int sizeE;
  long lTOTAL,lUNIQUE;
}GCLASS, *GClass;
GCLASS G;
//ローカル構造体
typedef struct{
  int BOUND1,BOUND2,TOPBIT,ENDBIT,SIDEMASK,LASTMASK;
  int mask;
  int aBoard[MAX];
  long COUNT2[MAX],COUNT4[MAX],COUNT8[MAX];
}local ;
//関数宣言
void symmetryOps(local *l);
//非再帰
void backTrack2D_NR(int y,int left,int down,int right,local *l);
void backTrack1D_NR(int y,int left,int down,int right,local *l);
//非再帰 通常版
void backTrack2_NR(int y,int left,int down,int right,local *l);
void backTrack1_NR(int y,int left,int down,int right,local *l);
//再帰
void backTrack2D(int y,int left,int down,int right,local *l);
void backTrack1D(int y,int left,int down,int right,local *l);
//再帰 通常版
void backTrack2(int y,int left,int down,int right,local *l);
void backTrack1(int y,int left,int down,int right,local *l);
//pthread
void *run(void *args);
void *NQueenThread();
void NQueen();
//
__global__ void cuda_kernel(
    int size,int mark,
    unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,
    unsigned int* results,int totalCond){
  const int tid=threadIdx.x;
  const int bid=blockIdx.x;
  const int idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10];
  __shared__ unsigned int left[THREAD_NUM][10];
  __shared__ unsigned int right[THREAD_NUM][10];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;
  unsigned int bit;
  if(idx<totalCond){
    down[tid][row]=totalDown[idx];
    left[tid][row]=totalLeft[idx];
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];
    while(row>=0){
      if((bitmap[tid][row]&mask)==mask){row--;}
      else{
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        if((bit&mask)!=0){
          if(row+1==mark){total++;row--;}
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{row--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
//
long long solve_nqueen_cuda(int size,int steps) {
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;
  unsigned int* leftCuda;
  unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}	
  total*=2;


  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
      >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] results;
  return total;
}
/** CUDA 初期化 **/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//
void symmetryOps(local *l){
  int own,ptn,you,bit;
  //90度回転
  if(l->aBoard[l->BOUND2]==1){ own=1; ptn=2;
    while(own<=G.sizeE){ bit=1; you=G.sizeE;
      while((l->aBoard[you]!=ptn)&&(l->aBoard[own]>=bit)){ bit<<=1; you--; }
      if(l->aBoard[own]>bit){ return; } if(l->aBoard[own]<bit){ break; }
      own++; ptn<<=1;
    }
    /** 90度回転して同型なら180度/270度回転も同型である */
    if(own>G.sizeE){ l->COUNT2[l->BOUND1]++; return; }
  }
  //180度回転
  if(l->aBoard[G.sizeE]==l->ENDBIT){ own=1; you=G.sizeE-1;
    while(own<=G.sizeE){ bit=1; ptn=l->TOPBIT;
      while((l->aBoard[you]!=ptn)&&(l->aBoard[own]>=bit)){ bit<<=1; ptn>>=1; }
      if(l->aBoard[own]>bit){ return; } if(l->aBoard[own]<bit){ break; }
      own++; you--;
    }
    /** 90度回転が同型でなくても180度回転が同型である事もある */
    if(own>G.sizeE){ l->COUNT4[l->BOUND1]++; return; }
  }
  //270度回転
  if(l->aBoard[l->BOUND1]==l->TOPBIT){ own=1; ptn=l->TOPBIT>>1;
    while(own<=G.sizeE){ bit=1; you=0;
      while((l->aBoard[you]!=ptn)&&(l->aBoard[own]>=bit)){ bit<<=1; you++; }
      if(l->aBoard[own]>bit){ return; } if(l->aBoard[own]<bit){ break; }
      own++; ptn>>=1;
    }
  }
  l->COUNT8[l->BOUND1]++;
}
//
//CPU 非再帰版 backTrack2//新しく記述
void backTrack2_NR(int row,int h_left,int h_down,int h_right,local *l){
    unsigned int left[G.size];
    unsigned int down[G.size];
    unsigned int right[G.size];
    unsigned int bitmap[G.size];
    left[row]=h_left;
    down[row]=h_down;
    right[row]=h_right;
    bitmap[row]=l->mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
	//【枝刈り】上部サイド枝刈り
	if(row<l->BOUND1){             	
	  bitmap[row]&=~l->SIDEMASK;
        //【枝刈り】下部サイド枝刈り
        }else if(row==l->BOUND2) {     	
          if((down[row]&l->SIDEMASK)==0){ row--; }
          if((down[row]&l->SIDEMASK)!=l->SIDEMASK){ bitmap[row]&=l->SIDEMASK; }
        }
        int save_bitmap=bitmap[row];
        bitmap[row]^=l->aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&l->mask)!=0){
          if(row==G.sizeE){
            if((save_bitmap&l->LASTMASK)==0){ 	
              symmetryOps(l);
              --row;
		    }
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=l->mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
//
//通常版 CPU 非再帰版 backTrack2
void backTrack2D_NR(int row,int left,int down,int right,local *l){
  int bitmap,bit;
  int b[100], *p=b;
  int odd=G.size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=G.size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(G.size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
mais1:bitmap=l->mask&~(left|down|right);
      // 【枝刈り】
      if(row==G.sizeE){
        if(bitmap){
          //【枝刈り】 最下段枝刈り
          if((bitmap&l->LASTMASK)==0){
            l->aBoard[row]=bitmap;
            symmetryOps(l);
          }
        }
      }else{
        //【枝刈り】上部サイド枝刈り
        if(row<l->BOUND1){
          bitmap&=~l->SIDEMASK;
          //【枝刈り】下部サイド枝刈り
        }else if(row==l->BOUND2){
          if(!(down&l->SIDEMASK))
            goto volta;
          if((down&l->SIDEMASK)!=l->SIDEMASK)
            bitmap&=l->SIDEMASK;
        }
        if(bitmap){
outro:bitmap^=l->aBoard[row]=bit=-bitmap&bitmap;
      if(bitmap){
        *p++=left;
        *p++=down;
        *p++=right;
      }
      *p++=bitmap;
      row++;
      left=(left|bit)<<1;
      down=down|bit;
      right=(right|bit)>>1;
      goto mais1;
      //Backtrack2(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
volta:if(p<=b)
        return;
      row--;
      bitmap=*--p;
      if(bitmap){
        right=*--p;
        down=*--p;
        left=*--p;
        goto outro;
      }else{
        goto volta;
      }
        }
      }
      goto volta;
  }
}
//CPU 非再帰版 backTrack
void backTrack1_NR(int row,int h_left,int h_down,int h_right,local *l){

    unsigned int left[G.size];
    unsigned int down[G.size];
    unsigned int right[G.size];
    unsigned int bitmap[G.size];
    left[row]=h_left;
    down[row]=h_down;
    right[row]=h_right;
    bitmap[row]=l->mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        if(row<l->BOUND1) {
          bitmap[row]&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
        }
        bitmap[row]^=l->aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&l->mask)!=0){
          if(row==G.sizeE){
            l->COUNT8[l->BOUND1]++;
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=l->mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }

}
//通常版 CPU 非再帰版 backTrack
void backTrack1D_NR(int row,int left,int down,int right,local *l){
  int bitmap,bit;
  int b[100], *p=b;
  int odd=G.size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=G.size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(G.size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
b1mais1:bitmap=l->mask&~(left|down|right);
        //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
        if(row==G.sizeE){
          if(bitmap){
            // l->aBoard[row]=bitmap;
            l->COUNT8[l->BOUND1]++;
          }
        }else{
          //【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい
          // ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
          if(row<l->BOUND1) {
            bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
          }
          if(bitmap){
b1outro:bitmap^=l->aBoard[row]=bit=-bitmap&bitmap;
        if(bitmap){
          *p++=left;
          *p++=down;
          *p++=right;
        }
        *p++=bitmap;
        row++;
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        goto b1mais1;
        //Backtrack1(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
b1volta:if(p<=b)
          return;
        row--;
        bitmap=*--p;
        if(bitmap){
          right=*--p;
          down=*--p;
          left=*--p;
          goto b1outro;
        }else{
          goto b1volta;
        }
          }
        }
        goto b1volta;
  }
}
//
//CPU 再帰版 backTrack
//ここに追記
void backTrack2(int row,int left,int down,int right,local *l){
 int bitmap=0;
 int bit=0;
 bitmap=(l->mask&~(left|down|right));
 if(row==G.sizeE){
   if(bitmap){
     //【枝刈り】 最下段枝刈り
     if((bitmap&l->LASTMASK)==0){ 	
       l->aBoard[row]=(-bitmap&bitmap);
       symmetryOps(l);
     }
   }
  }else{
    //【枝刈り】上部サイド枝刈り
    if(row<l->BOUND1){             	
      bitmap&=~l->SIDEMASK;
      //【枝刈り】下部サイド枝刈り
    }else if(row==l->BOUND2) {     	
      if((down&l->SIDEMASK)==0){ return; }
      if((down&l->SIDEMASK)!=l->SIDEMASK){ bitmap&=l->SIDEMASK; }
    }
    while(bitmap){
      bitmap^=l->aBoard[row]=bit=(-bitmap&bitmap);
      backTrack2(row+1,(left|bit)<<1, down|bit,(right|bit)>>1,l);
    }
  }

}
//通常版 CPU 再帰版 backTrack
void backTrack2D(int row,int left,int down,int right,local *l){
  int bit;
  int bitmap=l->mask&~(left|down|right);
  if(row==G.sizeE){ 								// 【枝刈り】
    if(bitmap){
      if((bitmap&l->LASTMASK)==0){ 	//【枝刈り】 最下段枝刈り
        l->aBoard[row]=bitmap;
        symmetryOps(l);
      }
    }
  }else{
    if(row<l->BOUND1){             	//【枝刈り】上部サイド枝刈り
      bitmap&=~l->SIDEMASK;
    }else if(row==l->BOUND2) {     	//【枝刈り】下部サイド枝刈り
      if((down&l->SIDEMASK)==0){ return; }
      if((down&l->SIDEMASK)!=l->SIDEMASK){ bitmap&=l->SIDEMASK; }
    }
    while(bitmap){
      bitmap^=l->aBoard[row]=bit=(-bitmap&bitmap);
      backTrack2D(row+1,(left|bit)<<1,down|bit,(right|bit)>>1,l);
    }
  }
}
//
//CPU 再帰版 backTrack
//ここに追記
void backTrack1(int row,int left,int down,int right,local *l){
 int bitmap=0;
 int bit=0;
 bitmap=(l->mask&~(left|down|right));
 if(row==G.sizeE){
   if(bitmap){
     l->COUNT8[l->BOUND1]++;
   }
  }else{
    if(row<l->BOUND1) {
      bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
    }
    while(bitmap){
      bitmap^=l->aBoard[row]=bit=(-bitmap&bitmap);
      backTrack1(row+1,(left|bit)<<1, down|bit,(right|bit)>>1,l);
    }
  }
}
//通常版 CPU 再帰版 backTrack
void backTrack1D(int row,int left,int down,int right,local *l){
  int bit;
  int bitmap=l->mask&~(left|down|right);
  //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
  if(row==G.sizeE) {
    if(bitmap){
      /* l->aBoard[row]=bitmap; */
      l->COUNT8[l->BOUND1]++;
    }
  }else{
    //【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい
    // ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
    if(row<l->BOUND1) {
      bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
    }
    while(bitmap){
      bitmap^=l->aBoard[row]=bit=(-bitmap&bitmap);
      backTrack1D(row+1,(left|bit)<<1,down|bit,(right|bit)>>1,l);
    }
  }
}
//
void *run(void *args){
  local *l=(local *)args;
  int bit=0;
  l->TOPBIT=1<<(G.size-1);
  // 最上段のクイーンが角にある場合の探索
  if(l->BOUND1>1 && l->BOUND1<G.sizeE) {
    int col=0;
    l->aBoard[0]=bit=(1<<0);
    int left=bit<<1;
    int down=bit;
    int right=bit>>1;
    if(l->BOUND1<G.sizeE) {
      // 角にクイーンを配置
      int col_j=l->BOUND1;
      l->aBoard[1]=bit=(1<<col_j);
      //２行目から探索
      if(NR==1){
        //非再帰
        backTrack1_NR(2,(left|bit)<<1,(down|bit),(right|bit)>>1,l);//GPU適用版
        //backTrack1D_NR(2,(left|bit)<<1,(down|bit),(right|bit)>>1,l);
      }else{
        //再帰
        backTrack1(2,(left|bit)<<1,(down|bit),(right|bit)>>1,l);//GPU適用版
        //backTrack1D(2,(left|bit)<<1,(down|bit),(right|bit)>>1,l);//通常版
      }
    }
  }
  l->ENDBIT=(l->TOPBIT>>l->BOUND1);
  l->SIDEMASK=l->LASTMASK=(l->TOPBIT|1);
  /* 最上段行のクイーンが角以外にある場合の探索
     ユニーク解に対する左右対称解を予め削除するには、
     左半分だけにクイーンを配置するようにすればよい */
  if(l->BOUND1>0&&l->BOUND2<G.sizeE&&l->BOUND1<l->BOUND2){
    for(int i=1; i<l->BOUND1; i++){
      l->LASTMASK=l->LASTMASK|l->LASTMASK>>1|l->LASTMASK<<1;
    }
    if(l->BOUND1<l->BOUND2) {
      int col=l->BOUND1;
      l->aBoard[0]=bit=(1<<col);
      if(NR==1){
        //printf("非再帰\n");
        backTrack2_NR(1,bit<<1,bit,bit>>1,l); //GPU適用版
        //backTrack2D_NR(1,bit<<1,bit,bit>>1,l);//通常版
      }else{
        //printf("再帰\n");
        backTrack2(1,bit<<1,bit,bit>>1,l); //GPU適用版
        //backTrack2D(1,bit<<1,bit,bit>>1,l);//通常版
      }
    }
    l->ENDBIT>>=G.size;
  }
  return 0;   //*run()の場合はreturn 0;が必要
}
//
void *NQueenThread(){
  local l[MAX];                //構造体 local型
  pthread_t pt[G.size];                 //スレッド childThread
  for(int BOUND1=G.sizeE,BOUND2=0;BOUND2<G.sizeE;BOUND1--,BOUND2++){
    l[BOUND1].mask=(1<<G.size)-1;
    l[BOUND1].BOUND1=BOUND1; l[BOUND1].BOUND2=BOUND2;         //B1 と B2を初期化
    for(int j=0;j<G.size;j++){ l[l->BOUND1].aBoard[j]=j; } // aB[]の初期化
    l[BOUND1].COUNT2[BOUND1]=l[BOUND1].COUNT4[BOUND1]=l[BOUND1].COUNT8[BOUND1]=0;//カウンターの初期化
    // チルドスレッドの生成
    int iFbRet=pthread_create(&pt[BOUND1],NULL,&run,&l[BOUND1]);
    if(iFbRet>0){
      printf("[mainThread] pthread_create #%d: %d\n", l[BOUND1].BOUND1, iFbRet);
    }
  }
  for(int BOUND1=G.sizeE,BOUND2=0;BOUND2<G.sizeE;BOUND1--,BOUND2++){
    pthread_join(pt[BOUND1],NULL);
  }
  //スレッド毎のカウンターを合計
  for(int BOUND1=G.sizeE,BOUND2=0;BOUND2<G.sizeE;BOUND1--,BOUND2++){
    G.lTOTAL+=l[BOUND1].COUNT2[BOUND1]*2+l[BOUND1].COUNT4[BOUND1]*4+l[BOUND1].COUNT8[BOUND1]*8;
    G.lUNIQUE+=l[BOUND1].COUNT2[BOUND1]+l[BOUND1].COUNT4[BOUND1]+l[BOUND1].COUNT8[BOUND1];
  }
  return 0;
}
//
void NQueen(){
  pthread_t pth;  //スレッド変数
  int iFbRet;
  // メインスレッドの生成
  // 拡張子 CUDA はpthreadをサポートしていませんので実行できません
  // コンパイルが通らないので 以下をコメントアウトします
  // Cディレクトリの 並列処理はC13_N-Queen.c を参考にして下さい。
  //iFbRet = pthread_create(&pth, NULL,&NQueenThread,NULL);
  if(iFbRet>0){
    printf("[main] pthread_create: %d\n", iFbRet); //エラー出力デバッグ用
  }
  pthread_join(pth,NULL); /* いちいちjoinをする */
}
//メインメソッド
int main(int argc,char** argv) {
  bool cpu=false,cpur=false,gpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r] n steps\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  if(cpu){
    printf("\n\n１３．CPU 非再帰 並列処理 pthread\n※nvccではpthreadは動きません！\n");
  }else if(cpur){
    printf("\n\n１３．CPUR 再帰 並列処理 pthread\n※nvccではpthreadは動きません！\n");
  }else if(gpu){
    printf("\n\n１３．GPU 非再帰 並列処理 CUDA\n");
  }
  if(cpu||cpur){
    printf("%s\n"," N:           Total           Unique          dd:hh:mm:ss.ms");
    struct timeval t0;
    struct timeval t1;
    int min=4; int targetN=18;
    for(int i=min;i<=targetN;i++){
      //TOTAL=0; UNIQUE=0;
      G.size=i; G.sizeE=i-1; //初期化
      G.lTOTAL=G.lUNIQUE=0;
      gettimeofday(&t0, NULL);
      //再帰
      if(cpur){
        //NR=0;NQueenD();
        NR=0;NQueen();
      }
      //非再帰
      if(cpu){ 
        //NR=1;NQueenD();
        NR=1;NQueen();
      }
      gettimeofday(&t1, NULL);
      int ss;int ms;int dd;
      if(t1.tv_usec<t0.tv_usec) {
        dd=(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      }else {
        dd=(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%16ld%17ld%12.2d:%02d:%02d:%02d.%02d\n", i,G.lTOTAL,G.lUNIQUE,dd,hh,mm,ss,ms);
    }
  }
  if(gpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=18;
    struct timeval t0;struct timeval t1;int ss;int ms;int dd;
    printf("%s\n"," N:          Total        Unique                 dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      Total=solve_nqueen_cuda(i,steps);
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%18ld%18ld%12.2d:%02d:%02d:%02d.%02d\n", i,Total,Unique,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
