
/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc CUDA**_N-Queen.cu && ./a.out (-c|-r|-g|-s)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 
                    -s SGPU(サマーズ版と思われる)


６．バックトラック＋ビットマップ

   ビット演算を使って高速化 状態をビットマップにパックし、処理する
   単純なバックトラックよりも２０〜３０倍高速
 
 　ビットマップであれば、シフトにより高速にデータを移動できる。
  フラグ配列ではデータの移動にO(N)の時間がかかるが、ビットマップであればO(1)
  フラグ配列のように、斜め方向に 2*N-1の要素を用意するのではなく、Nビットで充
  分。

 　配置可能なビット列を flags に入れ、-flags & flags で順にビットを取り出し処理。
 　バックトラックよりも２０−３０倍高速。
 
 ===================
 考え方 1
 ===================

 　Ｎ×ＮのチェスボードをＮ個のビットフィールドで表し、ひとつの横列の状態をひと
 つのビットフィールドに対応させます。(クイーンが置いてある位置のビットをONに
 する)
 　そしてバックトラッキングは0番目のビットフィールドから「下に向かって」順にい
 ずれかのビット位置をひとつだけONにして進めていきます。

 
 -----Q--    00000100 0番目のビットフィールド
 ---Q----    00010000 1番目のビットフィールド
 ------ Q-   00000010 2番目のビットフィールド
  Q-------   10000000 3番目のビットフィールド
 -------Q    00000001 4番目のビットフィールド
 -Q------    01000000 5番目のビットフィールド
 ---- Q---   00001000 6番目のビットフィールド
 -- Q-----   00100000 7番目のビットフィールド


 ===================
 考え方 2
 ===================

 次に、効き筋をチェックするためにさらに３つのビットフィールドを用意します。

 1. 左下に効き筋が進むもの: left 
 2. 真下に効き筋が進むもの: down
 3. 右下に効き筋が進むもの: right

次に、斜めの利き筋を考えます。
 上図の場合、
 1列目の右斜め上の利き筋は 3 番目(0x08)
 2列目の右斜め上の利き筋は 2 番目(0x04) になります。
 この値は 0 列目のクイーンの位置 0x10 を 1 ビットずつ「右シフト」すれば求める
 ことができます。
 また、左斜め上の利き筋の場合、1 列目では 5 番目(0x20) で 2 列目では 6 番目(0x40)
になるので、今度は 1 ビットずつ「左シフト」すれば求めることができます。

つまり、右シフトの利き筋を right、左シフトの利き筋を left で表すことで、クイー
ンの効き筋はrightとleftを1 ビットシフトするだけで求めることができるわけです。

  *-------------
 |. . . . . .
 |. . . -3. .  0x02 -|
 |. . -2. . .  0x04  |(1 bit 右シフト right)
 |. -1. . . .  0x08 -|
 |Q . . . . .  0x10 ←(Q の位置は 4   down)
 |. +1. . . .  0x20 -| 
 |. . +2. . .  0x40  |(1 bit 左シフト left)  
 |. . . +3. .  0x80 -|
  *-------------
  図：斜めの利き筋のチェック

 n番目のビットフィールドからn+1番目のビットフィールドに探索を進めるときに、そ
 の３つのビットフィールドとn番目のビットフィールド(bit)とのOR演算をそれぞれ行
 います。leftは左にひとつシフトし、downはそのまま、rightは右にひとつシフトして
 n+1番目のビットフィールド探索に渡してやります。

 left :(left |bit)<<1
 right:(right|bit)>>1
 down :   down|bit


 ===================
 考え方 3
 ===================

   n+1番目のビットフィールドの探索では、この３つのビットフィールドをOR演算した
 ビットフィールドを作り、それがONになっている位置は効き筋に当たるので置くことが
 できない位置ということになります。次にその３つのビットフィールドをORしたビッ
 トフィールドをビット反転させます。つまり「配置可能なビットがONになったビットフィー
 ルド」に変換します。そしてこの配置可能なビットフィールドを bitmap と呼ぶとして、
 次の演算を行なってみます。
 
 bit=-bitmap & bitmap;//一番右のビットを取り出す
 
   この演算式の意味を理解するには負の値がコンピュータにおける２進法ではどのよう
 に表現されているのかを知る必要があります。負の値を２進法で具体的に表わしてみる
 と次のようになります。
 
  00000011   3
  00000010   2
  00000001   1
  00000000   0
  11111111  -1
  11111110  -2
  11111101  -3
 
   正の値nを負の値-nにするときは、nをビット反転してから+1されています。そして、
 例えばn=22としてnと-nをAND演算すると下のようになります。nを２進法で表したときの
 一番下位のONビットがひとつだけ抽出される結果が得られるのです。極めて簡単な演算
 によって1ビット抽出を実現させていることが重要です。
 
      00010110   22
  AND 11101010  -22
 ------------------
      00000010
 
   さて、そこで下のようなwhile文を書けば、このループは bitmap のONビットの数の
 回数だけループすることになります。配置可能なパターンをひとつずつ全く無駄がなく
 生成されることになります。
 
 while(bitmap) {
     bit=-bitmap & bitmap;
     bitmap ^= bit;
     //ここでは配置可能なパターンがひとつずつ生成される(bit) 
 }

 実行結果
$ nvcc CUDA06_N-Queen.cu  && ./a.out -r
６．CPUR 再帰 バックトラック＋ビットマップ
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               0            0.00
 5:           10               0            0.00
 6:            4               0            0.00
 7:           40               0            0.00
 8:           92               0            0.00
 9:          352               0            0.00
10:          724               0            0.00
11:         2680               0            0.00
12:        14200               0            0.01
13:        73712               0            0.04
14:       365596               0            0.19
15:      2279184               0            1.24
16:     14772512               0            7.79
17:     95815104               0           57.57

$ nvcc CUDA06_N-Queen.cu  && ./a.out -c
６．CPU 非再帰 バックトラック＋ビットマップ
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               0            0.00
 5:           10               0            0.00
 6:            4               0            0.00
 7:           40               0            0.00
 8:           92               0            0.00
 9:          352               0            0.00
10:          724               0            0.00
11:         2680               0            0.00
12:        14200               0            0.01
13:        73712               0            0.04
14:       365596               0            0.21
15:      2279184               0            1.40
16:     14772512               0            8.78
17:     95815104               0         1:05.00

bash-3.2$ nvcc CUDA06_N-Queen.cu && ./a.out -s
６．SGPU 非再帰 バックトラック＋ビットマップ
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               0  00:00:00:00.05
 5:           10               0  00:00:00:00.00
 6:            4               0  00:00:00:00.00
 7:           40               0  00:00:00:00.00
 8:           92               0  00:00:00:00.00
 9:          352               0  00:00:00:00.02
10:          724               0  00:00:00:00.00
11:         2680               0  00:00:00:00.01
12:        14200               0  00:00:00:00.02
13:        73712               0  00:00:00:00.05
14:       365596               0  00:00:00:00.09
15:      2279184               0  00:00:00:00.49
16:     14772512               0  00:00:00:02.43
17:     95815104               0  00:00:00:18.44

$ nvcc CUDA06_N-Queen.cu  && ./a.out -g
６．GPU 非再帰 バックトラック＋ビットマップ
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               0  00:00:00:00.02
 5:           10               0  00:00:00:00.00
 6:            4               0  00:00:00:00.00
 7:           40               0  00:00:00:00.00
 8:           92               0  00:00:00:00.00
 9:          352               0  00:00:00:00.00
10:          724               0  00:00:00:00.00
11:         2680               0  00:00:00:00.02
12:        14200               0  00:00:00:00.07
13:        73712               0  00:00:00:00.10
14:       365596               0  00:00:00:00.10
15:      2279184               0  00:00:00:00.47
16:     14772512               0  00:00:00:02.37
17:     95815104               0  00:00:00:18.16

*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
//変数宣言
long TOTAL=0;         //CPU,CPUR
long UNIQUE=0;        //CPU,CPUR
//関数宣言 GPU
__global__ void cuda_kernel(int size,int mark,unsigned int* t_down,unsigned int* t_left,unsigned int* t_right,unsigned int* d_results,int totalCond);
long long solve_nqueen_cuda(int size,int steps);
void NQueenG(int size,int mask,int row,int steps);
//関数宣言 SGPU
__global__ void sgpu_cuda_kernel(int size,int mark,
		unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond);
long long sgpu_solve_nqueen_cuda(int size,int steps); 
//関数宣言 CPU
void TimeFormat(clock_t utime,char *form);
//関数宣言 CPU
void solve_nqueen(int size,int mask, int row,int* left,int* down,int* right,int* bitmap);
void NQueen(int size,int mask);
//関数宣言 CPUR
void solve_nqueenr(int size,int mask, int row,int left,int down,int right);
void NQueenR(int size,int mask,int row,int left,int down,int right);
//関数宣言 通常版
//  非再帰
void NQueenD(int size,int mask,int row);
//  再帰
void NQueenDR(int size,int mask,int row,int left,int down,int right);
//
//GPU
__global__ 
void cuda_kernel(
  int size,int mark,
  unsigned int* t_down,unsigned int* t_left,unsigned int* t_right,
  unsigned int* d_results,int totalCond){
  //threadIdx.x ブロック内のスレッドID,
  //blockIdx.x – グリッド内のブロックID,
  //blockDim.x – ブロックあたりのスレッドの数
  const int tid=threadIdx.x;//ブロック内のスレッドID
  const int bid=blockIdx.x;//グリッド内のブロックID
  const int idx=bid*blockDim.x+tid;//全体通してのID
  __shared__ unsigned int down[THREAD_NUM][10];//sharedメモリを使う ブロック内スレッドで共有
  __shared__ unsigned int left[THREAD_NUM][10];//THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int right[THREAD_NUM][10];//10で固定なのは現在のmaskの設定でGPUで実行するのは最大10だから
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;//row=0となってるが1行目からやっているわけではなくmask行目以降からスタート n=8 なら mask==2 なので そこからスタート
  unsigned int bit;
  if(idx<totalCond){//余分なスレッドは動かさない GPUはsteps数起動するがtotalCond以上は空回しする
  //printf("size:%d,mark:%d:t_down:%d:t_left:%d:t_right:%d,totalCond:%d\n",size,mark,t_down[idx],t_left[idx],t_right[idx],totalCond);
    down[tid][row]=t_down[idx];//t_down,t_left,t_rightの情報をdown,left,rightに詰め直す 
    left[tid][row]=t_left[idx];//CPU で詰め込んだ t_はsteps個あるがブロック内ではブロックあたりのスレッドすうに限定されるので idxでよい
    right[tid][row]=t_right[idx];
    bitmap[tid][row]=mask&~(down[tid][row]|left[tid][row]|right[tid][row]);//down,left,rightからbitmapを出す
    while(row>=0){
      if(bitmap[tid][row]==0){///bitmap[tid][row]=00000000 クイーンをどこにも置けないので1行上に戻る
        --row;
      }else{
        bitmap[tid][row]^=bit=(-bitmap[tid][row]&bitmap[tid][row]); //クイーンを置く
        if((bit&mask)!=0){//置く場所があるかどうか
          //printf("row:%d:mark:%d\n",row,mark);
          if(row+1==mark){//最終行?最終行から１個前の行まで無事到達したら 加算する
            total++;
            //printf("total:%d\n",total);
            --row;
          }else{
            int n=row++;//クイーン置いた位置から次の行へ渡すdown,left,right,bitmapを出す
            down[tid][row]=down[tid][n]|bit;
            left[tid][row]=(left[tid][n]|bit)<<1;
            right[tid][row]=(right[tid][n]|bit)>>1;
            bitmap[tid][row]=mask&~(down[tid][row]|left[tid][row]|right[tid][row]);
          }
        }else{//置く場所がなければ１個上に
          --row;
        }
      }
    }
    sum[tid]=total;//最後sum[tid]に加算する
  }else{//totalCond未満は空回しするので当然 totalは加算しない
      sum[tid]=0;
  } 
  //__syncthreads()で、ブロック内のスレッド間の同期をとれます。
  //同期を取るということは、全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} //__syncthreads();は複数個必要1個だけ記述したら数が違った
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){d_results[bid]=sum[0];}
  //__syncthreads();//これだとn13以降数が合わない
  //for (int k = 0; k < THREAD_NUM; ++k){
  //  d_results[bid]+=sum[k];
  //}
  //__syncthreads();
}
//
// GPU
long solve_nqueen_cuda(int size,int mask,int row,int n_left,int n_down,int n_right,int steps) {//NQueenに相当
  register int bitmap[32];//bitmapを配列で持つことによりstackを使わないで1行前に戻れる
  register int bit;

  //host
  register int h_down[size];
  hipHostMalloc((void**) &h_down,sizeof(int)*steps, hipHostMallocDefault);
  register int h_right[size];
  hipHostMalloc((void**) &h_right,sizeof(int)*steps, hipHostMallocDefault);
  register int h_left[size];
  hipHostMalloc((void**) &h_left,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_down=new unsigned int[steps];
  hipHostMalloc((void**) &t_down,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_left=new unsigned int[steps];
  hipHostMalloc((void**) &t_left,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_right=new unsigned int[steps];
  hipHostMalloc((void**) &t_right,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_results=new unsigned int[steps];
  hipHostMalloc((void**) &h_results,sizeof(int)*steps/THREAD_NUM, hipHostMallocDefault);

  //device
  unsigned int* d_down;
  hipMalloc((void**) &d_down,sizeof(int)*steps);
  unsigned int* d_left;
  hipMalloc((void**) &d_left,sizeof(int)*steps);
  unsigned int* d_right;
  hipMalloc((void**) &d_right,sizeof(int)*steps);
  unsigned int* d_results;
  hipMalloc((void**) &d_results,sizeof(int)*steps/THREAD_NUM);

  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  const unsigned int mark=size>11?size-10:2;
  const unsigned int h_mark=row;

  h_left[row]=n_left;
  h_down[row]=n_down;
  h_right[row]=n_right;
  bitmap[row]=mask&~(h_left[row]|h_down[row]|h_right[row]);

  //12行目までは3行目までCPU->row==mark以下で 3行目までのdown,left,right情報を t_down,t_left,t_rightに格納する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く　例えば n15だとrow=5までCPUで実行し、それ以降はGPU(現在の設定だとGPUでは最大10行実行するようになっている)
  long total=0;
  int totalCond=0;
  //bit=0;
  //h_down[0]=h_left[0]=h_right[0]=0;
  bool matched=false;
  while(row>=h_mark){
    if(bitmap[row]==0){//bitmap[row]=00000000 クイーンをどこにも置けないので1行上に戻る
        row--;
    }else{//おける場所があれば進む
        bitmap[row]^=bit=(-bitmap[row]&bitmap[row]); //クイーンを置く
        if((bit&mask)!=0){//置く場所があれば先に進む
          int n=row++;//クイーン置いた位置から次の行へ渡す down,left,right,bitmapを出す
          h_down[row]=h_down[n]|bit;
          h_left[row]=(h_left[n]|bit)<<1;
          h_right[row]=(h_right[n]|bit)>>1;
          bitmap[row]=mask&~(h_down[row]|h_left[row]|h_right[row]);
          if(row==mark){
            //3行目(mark)にクイーンを１個ずつ置いていって、down,left,right情報を格納、
            //その次の行へは進まない。その行で可能な場所にクイーン置き終わったらGPU並列実行
            t_down[totalCond]=h_down[row];//totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
            t_left[totalCond]=h_left[row];//row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報をt_down,t_left,t_rightに格納する
            t_right[totalCond]=h_right[row];
            totalCond++;//スレッド数をインクリメントする
            //printf("totalCond:%d",totalCond);
            //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同時並行稼働数を制御
            //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増えて行くと超えるようになる。
            if(totalCond==steps){//ここではtotalCond==stepsの場合だけこの中へ
              if(matched){//matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
                hipMemcpy(h_results,d_results,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=h_results[col];}
                matched=false;
              }
              hipMemcpy(d_down,t_down,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(d_left,t_left,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(d_right,t_right,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,d_down,d_left,d_right,d_results,totalCond);//size-mark は何行GPUを実行するか totalCondはスレッド数
              //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのはtotalCondの数だけでそれ以外は空回しになる
              matched=true;//GPU内でカウントしているので、GPUから出たらmatched=trueになってる
              totalCond=0;//totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始する(これによりなんどもsteps数分だけGPUを起動できる)
              
            }
            --row;//t_down,t_left,t_rightに情報を格納したら1行上に上がる
            //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いてt_down,t_left,t_rightに情報を格納する
          }
        }else{//置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通にnqueenをやる
          --row;
        }
      }
    }
   if(matched){//matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
               hipMemcpy(h_results,d_results,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
               for(int col=0;col<steps/THREAD_NUM;col++){total+=h_results[col];}
               matched=false;}
               hipMemcpy(d_down,t_down,sizeof(int)*totalCond,hipMemcpyHostToDevice);
hipMemcpy(d_left,t_left,sizeof(int)*totalCond,hipMemcpyHostToDevice);
hipMemcpy(d_right,t_right,sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
   cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
   >>>(size,size-mark,d_down,d_left,d_right,d_results,totalCond);//size-mark は何行GPUを実行するか totalCondはスレッド数
   //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのはtotalCondの数だけでそれ以外は空回しになる
    hipMemcpy(h_results,d_results,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=h_results[col];}
    
   hipFree(d_down);
   hipFree(d_left);
   hipFree(d_right);
   hipFree(d_results);
   hipHostFree(t_down);
   hipHostFree(t_left);
   hipHostFree(t_right);
   hipHostFree(h_down);
   hipHostFree(h_left);
   hipHostFree(h_right);
   hipHostFree(h_results);
   //printf("total:%ld\n",total);
   return total;
}
//
//GPU
void NQueenG(int size,int steps){
  register int sizeE=size-1;
  register int bit;
  register int mask=((1<<size)-1);
  long total;
  if(size<=0||size>32){return;}
  bit=0;
  //bitmap[0]=mask;
  //down[0]=left[0]=right[0]=0;
  //偶数、奇数共通
  for(int col=0;col<size/2;col++){//右側半分だけクイーンを置く
    bit=(1<<col);//
    //down[1]=bit;//再帰の場合は down,left,right,bitmapは現在の行だけで>良いが
    //eft[1]=bit<<1;//非再帰の場合は全行情報を配列に入れて行の上がり下がりをする
    //right[1]=bit>>1;
    //bitmap[1]=mask&~(left[1]|down[1]|right[1]);
    //solve_nqueen(size,mask,1,left,down,right);
    total=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
    TOTAL+=total;
  }
  TOTAL=TOTAL*2;//ミラーなのでTOTALを２倍する
  //奇数の場合はさらに中央にクイーンを置く
  if(size%2==1){
    bit=(1<<(sizeE)/2);
    //down[1]=bit;
    //left[1]=bit<<1;
    //right[1]=bit>>1;
    //bitmap[1]=mask&~(left[1]|down[1]|right[1]);
    //solve_nqueen(size,mask,1,left,down,right);
    total=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
    TOTAL+=total;  
  }
}
//
//SGPU
__global__ 
void sgpu_cuda_kernel(
    int size,int mark,
    unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,
    unsigned int* results,int totalCond){
  const int tid=threadIdx.x;
  const int bid=blockIdx.x;
  const int idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10];
  __shared__ unsigned int left[THREAD_NUM][10];
  __shared__ unsigned int right[THREAD_NUM][10];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;
  unsigned int bit;
  if(idx<totalCond){
    down[tid][row]=totalDown[idx];
    left[tid][row]=totalLeft[idx];
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];
    while(row>=0){
      if((bitmap[tid][row]&mask)==mask){row--;}
      else{
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        if((bit&mask)!=0){
          if(row+1==mark){total++;row--;}
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{row--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
//
//SGPU
long long sgpu_solve_nqueen_cuda(int size,int steps) {
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;
  unsigned int* leftCuda;
  unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}	
  total*=2;


  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
      >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] results;
  return total;
}
//
//CUDA 初期化
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form){
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//
//CPU 非再帰版 ロジックメソッド
void solve_nqueen(int size,int mask, int row,int* left,int* down,int* right,int* bitmap){
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        bitmap[row]^=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            TOTAL++;
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
//
//非再帰版
void NQueen(int size,int mask){
  register int sizeE=size-1;
  register int bitmap[size];
  register int down[size],right[size],left[size];
  register int bit;
  if(size<=0||size>32){return;}
  bit=0;
  bitmap[0]=mask;
  down[0]=left[0]=right[0]=0;
  //偶数、奇数共通
  for(int col=0;col<size/2;col++){//右側半分だけクイーンを置く
    bit=(1<<col);//
    down[1]=bit;//再帰の場合は down,left,right,bitmapは現在の行だけで良いが
    left[1]=bit<<1;//非再帰の場合は全行情報を配列に入れて行の上がり下がりをする
    right[1]=bit>>1;
    bitmap[1]=mask&~(left[1]|down[1]|right[1]);
    solve_nqueen(size,mask,1,left,down,right,bitmap);
  }
  TOTAL*=2;//ミラーなのでTOTALを２倍する
  //奇数の場合はさらに中央にクイーンを置く
  if(size%2==1){
    bit=(1<<(sizeE)/2);
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=mask&~(left[1]|down[1]|right[1]);
    solve_nqueen(size,mask,1,left,down,right,bitmap);
  }  
}
//
//CPUR 再帰版 ロジックメソッド
void solve_nqueenr(int size,int mask, int row,int left,int down,int right){
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
    if(bitmap){
      TOTAL++;
    }
  }else{
    while(bitmap){
      bitmap^=bit=(-bitmap&bitmap);
      solve_nqueenr(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//
//CPUR 再帰版 ロジックメソッド
void NQueenR(int size,int mask, int row,int left,int down,int right){
  int bit=0;
  int sizeE=size-1;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    solve_nqueenr(size,mask,1,bit<<1,bit,bit>>1);
  }
  TOTAL*=2;
  if(size%2==1){
    bit=(1<<(sizeE)/2);
    solve_nqueenr(size,mask,1,bit<<1,bit,bit>>1);
  }
}
//
//通常版 CPU 非再帰版 ロジックメソッド
void NQueenD(int size,int mask,int row){
  int aStack[size];
  int* pnStack;
  int bit;
  int bitmap;
  int sizeE=size-1;
  int down[size],right[size],left[size];
  aStack[0]=-1; 
  pnStack=aStack+1;
  bit=0;
  bitmap=mask;
  down[0]=left[0]=right[0]=0;
  while(true){
    if(bitmap){
      bitmap^=bit=(-bitmap&bitmap); 
      if(row==sizeE){
        TOTAL++;
        bitmap=*--pnStack;
        --row;
        continue;
      }else{
        int n=row++;
        left[row]=(left[n]|bit)<<1;
        down[row]=down[n]|bit;
        right[row]=(right[n]|bit)>>1;
        *pnStack++=bitmap;
        bitmap=mask&~(left[row]|down[row]|right[row]);
        continue;
      }
    }else{ 
      bitmap=*--pnStack;
      if(pnStack==aStack){ break ; }
      --row;
      continue;
    }
  }
}
//
//通常版 CPUR 再帰版　ロジックメソッド
void NQueenDR(int size,int mask,int row,int left,int down,int right){
  int bitmap=0;
  int bit=0;
  if(row==size){
    TOTAL++;
  }else{
    bitmap=(mask&~(left|down|right));
    while(bitmap){
      bitmap^=bit=(-bitmap&bitmap);
      NQueenDR(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//
//メインメソッド
int main(int argc,char** argv) {
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  if(cpu){
    printf("\n\n６．CPU 非再帰 バックトラック＋ビットマップ\n");
  }else if(cpur){
    printf("\n\n６．CPUR 再帰 バックトラック＋ビットマップ\n");
  }else if(gpu){
    printf("\n\n６．GPU 非再帰 バックトラック＋ビットマップ\n");
  }else if(sgpu){
    printf("\n\n６．SGPU 非再帰 バックトラック＋ビットマップ\n");
  }
  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;          //速度計測用
    char t[20];          //hh:mm:ss.msを格納
    int min=4;
    int targetN=17;
    int mask;
    for(int i=min;i<=targetN;i++){
      TOTAL=0;
      UNIQUE=0;
      mask=((1<<i)-1);
      st=clock();
      //
      //CPUR
      if(cpur){ 
        NQueenR(i,mask,0,0,0,0); 
        //NQueenDR(i,mask,0,0,0,0);//通常版
      }
      //CPU
      if(cpu){ 
        NQueen(i,mask); 
        //NQueenD(i,mask,0); //通常版
      }
      //
      TimeFormat(clock()-st,t);
      printf("%2d:%13ld%16ld%s\n",i,TOTAL,UNIQUE,t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=17;
    struct timeval t0;struct timeval t1;
    int ss;int ms;int dd;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);  // 計測開始
      if(gpu){
        TOTAL=0;
        UNIQUE=0;
        NQueenG(i,steps);
      }else if(sgpu){
        TOTAL=sgpu_solve_nqueen_cuda(i,steps);
      	UNIQUE=0;
      }
      gettimeofday(&t1,NULL);  // 計測終了
      if(t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
