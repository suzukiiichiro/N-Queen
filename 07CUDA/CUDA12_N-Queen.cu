
/**
 Cで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 実行
 $ gcc -Wall -W -O3 -g -ftrapv -std=c99 GCC12.c && ./a.out [-c|-r]


 １２．対称解除法の最適化


bash-3.2$ gcc -Wall -W -O3 -g -ftrapv -std=c99 -pthread GCC12.c && ./a.out -r
１２．CPUR 再帰 対称解除法の最適化
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.00
12:        14200            1787            0.00
13:        73712            9233            0.01
14:       365596           45752            0.07
15:      2279184          285053            0.40
16:     14772512         1846955            2.61
17:     95815104        11977939           18.05


bash-3.2$ gcc -Wall -W -O3 -g -ftrapv -std=c99 -pthread GCC12.c && ./a.out -c
１２．CPU 非再帰 対称解除法の最適化
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.00
12:        14200            1787            0.00
13:        73712            9233            0.01
14:       365596           45752            0.06
15:      2279184          285053            0.34
16:     14772512         1846955            2.24
17:     95815104        11977939           15.72

bash-3.2$ nvcc CUDA12_N-Queen.cu && ./a.out -g
１２．GPU 非再帰 枝刈り
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               1  00:00:00:00.03
 5:           10               2  00:00:00:00.00
 6:            4               1  00:00:00:00.01
 7:           40               6  00:00:00:00.01
 8:           92              12  00:00:00:00.01
 9:          352              46  00:00:00:00.02
10:          724              92  00:00:00:00.02
11:         2680             341  00:00:00:00.03
12:        14200            1787  00:00:00:00.04
13:        73712            9233  00:00:00:00.06
14:       365596           45752  00:00:00:00.18
15:      2279184          285053  00:00:00:00.86
16:     14772512         1846955  00:00:00:04.47
17:     95815104        11977939  00:00:00:32.45
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#include <unistd.h>

#define THREAD_NUM		96
#define MAX 27
//変数宣言
int down[2*MAX-1];  //CPU down:flagA 縦 配置フラグ　
int left[2*MAX-1];  //CPU left:flagB 斜め配置フラグ　
int right[2*MAX-1]; //CPU right:flagC 斜め配置フラグ　
unsigned int aBoard[MAX];
int aT[MAX];
int aS[MAX];
long TOTAL=0;
long UNIQUE=0;
int COUNT2,COUNT4,COUNT8;
int BOUND1,BOUND2,TOPBIT,ENDBIT,SIDEMASK,LASTMASK;
//関数宣言 GPU
__global__ void cuda_kernel(int size,int mark,unsigned int* t_down,unsigned int* t_left,unsigned int* t_right,unsigned int* d_results,int totalCond);
long long solve_nqueen_cuda(int size,int steps);
void NQueenG(int size,int mask,int row,int steps);
__device__ int symmetryOps_bitmap_gpu(int si,unsigned int *d_aBoard,int *d_aT,int *d_aS);
//関数宣言 GPU
bool InitCUDA();
//関数宣言 SGPU
__global__ void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond);
long long sgpu_solve_nqueen_cuda(int size,int steps);
//関数宣言 CPU/GPU
__device__ __host__ void rotate_bitmap(int bf[],int af[],int si);
__device__ __host__ void vMirror_bitmap(int bf[],int af[],int si);
__device__ __host__ int intncmp(int lt[],int rt[],int n);
__device__ __host__ int rh(int a,int size);
//関数宣言
void TimeFormat(clock_t utime,char *form);
long getUnique();
long getTotal();
void symmetryOps_bitmap(int si);
//関数宣言 CPU
void solve_nqueen(int size,int mask, int row,int* left,int* down,int* right,int* bitmap);
void NQueen(int size,int mask);
//関数宣言 CPUR
void solve_nqueenr(int size,int mask, int row,int left,int down,int right);
void NQueenR(int size,int mask);
//関数宣言 通常版
void NQueenD(int size,int mask);
void NQueenDR(int size,int mask,int row,int left,int down,int right,int ex1,int ex2);
//
//
__global__ void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond){
  const int tid=threadIdx.x;
  const int bid=blockIdx.x;
  const int idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10];
  __shared__ unsigned int left[THREAD_NUM][10];
  __shared__ unsigned int right[THREAD_NUM][10];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;
  unsigned int bit;
  if(idx<totalCond){
    down[tid][row]=totalDown[idx];
    left[tid][row]=totalLeft[idx];
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];
    while(row>=0){
      if((bitmap[tid][row]&mask)==mask){row--;}
      else{
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        if((bit&mask)!=0){
          if(row+1==mark){total++;row--;}
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{row--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
//
long long sgpu_solve_nqueen_cuda(int size,int steps) {
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;
  unsigned int* leftCuda;
  unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}	
  total*=2;


  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
      >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] results;
  return total;
}
//
/** CUDA 初期化 **/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form){
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//
long getUnique(){
  return COUNT2+COUNT4+COUNT8;
}
//
long getTotal(){
  return COUNT2*2+COUNT4*4+COUNT8*8;
}
//
void symmetryOps(int si){
  int own,ptn,you,bit;
  //90度回転
  if(aBoard[BOUND2]==1){ own=1; ptn=2;
    while(own<=si-1){ bit=1; you=si-1;
      while((aBoard[you]!=ptn)&&(aBoard[own]>=bit)){ bit<<=1; you--; }
      if(aBoard[own]>bit){ return; } if(aBoard[own]<bit){ break; }
      own++; ptn<<=1;
    }
    /** 90度回転して同型なら180度/270度回転も同型である */
    if(own>si-1){ COUNT2++; return; }
  }
  //180度回転
  if(aBoard[si-1]==ENDBIT){ own=1; you=si-1-1;
    while(own<=si-1){ bit=1; ptn=TOPBIT;
      while((aBoard[you]!=ptn)&&(aBoard[own]>=bit)){ bit<<=1; ptn>>=1; }
      if(aBoard[own]>bit){ return; } if(aBoard[own]<bit){ break; }
      own++; you--;
    }
    /** 90度回転が同型でなくても180度回転が同型である事もある */
    if(own>si-1){ COUNT4++; return; }
  }
  //270度回転
  if(aBoard[BOUND1]==TOPBIT){ own=1; ptn=TOPBIT>>1;
    while(own<=si-1){ bit=1; you=0;
      while((aBoard[you]!=ptn)&&(aBoard[own]>=bit)){ bit<<=1; you++; }
      if(aBoard[own]>bit){ return; } if(aBoard[own]<bit){ break; }
      own++; ptn>>=1;
    }
  }
  COUNT8++;
}

//GPU
__device__
int symmetryOps_gpu(int si,unsigned int *d_aBoard,int BOUND1,int BOUND2,int TOPBIT,int ENDBIT){
      int own,ptn,you,bit;
  //90度回転
  if(d_aBoard[BOUND2]==1){ own=1; ptn=2;
    while(own<=si-1){ bit=1; you=si-1;
      while((d_aBoard[you]!=ptn)&&(d_aBoard[own]>=bit)){ bit<<=1; you--; }
      if(d_aBoard[own]>bit){ return 0; } if(d_aBoard[own]<bit){ break; }
      own++; ptn<<=1;
    }
    /** 90度回転して同型なら180度/270度回転も同型である */
    if(own>si-1){ return 2; }
  }
  //180度回転
  if(d_aBoard[si-1]==ENDBIT){ own=1; you=si-1-1;
    while(own<=si-1){ bit=1; ptn=TOPBIT;
      while((d_aBoard[you]!=ptn)&&(d_aBoard[own]>=bit)){ bit<<=1; ptn>>=1; }
      if(d_aBoard[own]>bit){ return 0; } if(d_aBoard[own]<bit){ break; }
      own++; you--;
    }
    /** 90度回転が同型でなくても180度回転が同型である事もある */
    if(own>si-1){ return 4; }
  }
  //270度回転
  if(d_aBoard[BOUND1]==TOPBIT){ own=1; ptn=TOPBIT>>1;
    while(own<=si-1){ bit=1; you=0;
      while((d_aBoard[you]!=ptn)&&(d_aBoard[own]>=bit)){ bit<<=1; you++; }
      if(d_aBoard[own]>bit){ return 0; } if(d_aBoard[own]<bit){ break; }
      own++; ptn>>=1;
    }
  }
  return 8; 

}

// GPU
__global__ 
void cuda_kernel_b1(
    int size,int mark,
    unsigned int* t_down,unsigned int* t_left,unsigned int* t_right,
    unsigned int* d_total,unsigned int* d_uniq,unsigned int* t_aBoard,int totalCond,int h_row,int BOUND1){
     //threadIdx.x ブロック内のスレッドID,blockIdx.x – グリッド内のブロックID,blockDim.x – ブロックあたりのスレッドの数
  const int tid=threadIdx.x;//ブロック内のスレッドID
  const int bid=blockIdx.x;//グリッド内のブロックID
  const int idx=bid*blockDim.x+tid;//全体通してのID
  __shared__ unsigned int down[THREAD_NUM][10];//sharedメモリを使う ブロック内スレッドで共有
  __shared__ unsigned int left[THREAD_NUM][10];//THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int right[THREAD_NUM][10];//10で固定なのは現在のmaskの設定でGPUで実行するのは最大10だから
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  __shared__ unsigned int usum[THREAD_NUM];
  
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int unique=0;
  int row=0;//row=0となってるが1行目からやっているわけではなくmask行目以降からスタート n=8 なら mask==2 なので そこからスタート
  unsigned int bit;
  if(idx<totalCond){//余分なスレッドは動かさない GPUはsteps数起動するがtotalCond以上は空回しする
    down[tid][row]=t_down[idx];//t_down,t_left,t_rightの情報をdown,left,rightに詰め直す 
    left[tid][row]=t_left[idx];//CPU で詰め込んだ t_はsteps個あるがブロック内ではブロックあたりのスレッドすうに限定されるので idxでよい
    right[tid][row]=t_right[idx];
    
    bitmap[tid][row]=mask&~(down[tid][row]|left[tid][row]|right[tid][row]);//down,left,rightからbitmapを出す
    while(row>=0){
      if(bitmap[tid][row]==0){///bitmap[tid][row]=00000000 クイーンをどこにも置けないので1行上に戻る
        --row;
      }else{
        if(row+h_row<BOUND1) {
          bitmap[tid][row]&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
        }  
        bitmap[tid][row]^=bit=(-bitmap[tid][row]&bitmap[tid][row]); //クイーンを置く
        
        if((bit&mask)!=0){//置く場所があるかどうか
          if(row+1==mark){//最終行?最終行から１個前の行まで無事到達したら 加算する
        
            int s=8; 
            if(s!=0){
            //print(size); //print()でTOTALを++しない
            //ホストに戻す配列にTOTALを入れる
            //スレッドが１つの場合は配列は１個
              unique++; 
              total+=s;   //対称解除で得られた解数を加算
            }
            --row;
          }else{
            int n=row++;//クイーン置いた位置から次の行へ渡すdown,left,right,bitmapを出す
            down[tid][row]=down[tid][n]|bit;
            left[tid][row]=(left[tid][n]|bit)<<1;
            right[tid][row]=(right[tid][n]|bit)>>1;
            bitmap[tid][row]=mask&~(down[tid][row]|left[tid][row]|right[tid][row]);
          }
        }else{//置く場所がなければ１個上に
            --row;
        }
      }
    }
    sum[tid]=total;//最後sum[tid]に加算する
    usum[tid]=unique;
  }else{//totalCond未満は空回しするので当然 totalは加算しない
      sum[tid]=0;
      usum[tid]=0;
      } 
  //__syncthreads()で、ブロック内のスレッド間の同期をとれます。
  //同期を取るということは、全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];usum[tid]+=usum[tid+64];} //__syncthreads();は複数個必要1個だけ記述したら数が違った
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];usum[tid]+=usum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];usum[tid]+=usum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];usum[tid]+=usum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];usum[tid]+=usum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];usum[tid]+=usum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];usum[tid]+=usum[tid+1];} 
  __syncthreads();if(tid==0){d_total[bid]=sum[0];d_uniq[bid]=usum[0];}
  //__syncthreads();//これだとn13以降数が合わない
  //for (int k = 0; k < THREAD_NUM; ++k){
  //  d_total[bid]+=sum[k];
  //  d_uniq[bid]+=usum[k];
  //}
  //__syncthreads();

}
//
__global__ 
void cuda_kernel_b2(
    int size,int mark,
    unsigned int* t_down,unsigned int* t_left,unsigned int* t_right,
    unsigned int* d_total,unsigned int* d_uniq,unsigned int* t_aBoard,int totalCond,int h_row,int B1,int B2,int SM,int LM,int TB,int EB){
     //threadIdx.x ブロック内のスレッドID,blockIdx.x – グリッド内のブロックID,blockDim.x – ブロックあたりのスレッドの数
  const int tid=threadIdx.x;//ブロック内のスレッドID
  const int bid=blockIdx.x;//グリッド内のブロックID
  const int idx=bid*blockDim.x+tid;//全体通してのID
  __shared__ unsigned int down[THREAD_NUM][10];//sharedメモリを使う ブロック内スレッドで共有
  __shared__ unsigned int left[THREAD_NUM][10];//THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int right[THREAD_NUM][10];//10で固定なのは現在のmaskの設定でGPUで実行するのは最大10だから
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int c_aBoard[THREAD_NUM][MAX];
  __shared__ unsigned int sum[THREAD_NUM];
  __shared__ unsigned int usum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int unique=0;
  int row=0;//row=0となってるが1行目からやっているわけではなくmask行目以降からスタート n=8 なら mask==2 なので そこからスタート
  unsigned int bit;
  if(idx<totalCond){//余分なスレッドは動かさない GPUはsteps数起動するがtotalCond以上は空回しする
    //printf("cuda:start\n");
  
    down[tid][row]=t_down[idx];//t_down,t_left,t_rightの情報をdown,left,rightに詰め直す 
    left[tid][row]=t_left[idx];//CPU で詰め込んだ t_はsteps個あるがブロック内ではブロックあたりのスレッドすうに限定されるので idxでよい
    right[tid][row]=t_right[idx];
    for(int i=0;i<size;i++){
    
      //c_aBoard[tid][i]=t_aBoard[idx][i];   
      c_aBoard[tid][i]=t_aBoard[idx*MAX+i]; //２次元配列だが1次元的に利用
      
    }
 
    bitmap[tid][row]=mask&~(down[tid][row]|left[tid][row]|right[tid][row]);//down,left,rightからbitmapを出す
    while(row>=0){
      if(bitmap[tid][row]==0){///bitmap[tid][row]=00000000 クイーンをどこにも置けないので1行上に戻る
        --row;
      }else{
         //printf("row:%d:bit:%d\n",row,bitmap[tid][row]);
         //【枝刈り】上部サイド枝刈り
	       if(row+h_row<B1){             	
             //printf("BOUND1_row:%d:h_row:%d:row+hrow:%d:bit:%d\n",row,h_row,row+h_row,bitmap[tid][row]);
 	         bitmap[tid][row]&=~SM;
        //【枝刈り】下部サイド枝刈り
        }else if(row+h_row==B2) {     	
            //printf("BOUND2_row:%d:h_row:%d:row+hrow:%d:bit:%d\n",row,h_row,row+h_row,bitmap[tid][row]);
            if((down[tid][row]&SM)==0){ 
               row--; 
               //printf("BOUND2_row\n");
            }
            if((down[tid][row]&SM)!=SM){ 
              bitmap[tid][row]&=SM; 
              //printf("BOUND2_SIDEMASK:%d\n",SM);            
            }
 
        }
        int save_bitmap=bitmap[tid][row];
        bitmap[tid][row]^=c_aBoard[tid][row+h_row]=bit=(-bitmap[tid][row]&bitmap[tid][row]); //クイーンを置く
        
        if((bit&mask)!=0){//置く場所があるかどうか
          if(row+1==mark){//最終行?最終行から１個前の行まで無事到達したら 加算する
            if((save_bitmap&LM)==0){  
              int s=symmetryOps_gpu(size,c_aBoard[tid],B1,B2,TB,EB); 
              //printf("row:%d:bit:%d:s:%d\n",row,bitmap[tid][row],s);
              if(s!=0){
              //print(size); //print()でTOTALを++しない
              //ホストに戻す配列にTOTALを入れる
              //スレッドが１つの場合は配列は１個
                unique++; 
                total+=s;   //対称解除で得られた解数を加算
              }
              --row;
            }
          }else{
            int n=row++;//クイーン置いた位置から次の行へ渡すdown,left,right,bitmapを出す
            down[tid][row]=down[tid][n]|bit;
            left[tid][row]=(left[tid][n]|bit)<<1;
            right[tid][row]=(right[tid][n]|bit)>>1;
            bitmap[tid][row]=mask&~(down[tid][row]|left[tid][row]|right[tid][row]);
          }
        }else{//置く場所がなければ１個上に
            --row;
        }
      }
    }
    sum[tid]=total;//最後sum[tid]に加算する
    usum[tid]=unique;
  }else{//totalCond未満は空回しするので当然 totalは加算しない
      sum[tid]=0;
      usum[tid]=0;
      } 
  //__syncthreads()で、ブロック内のスレッド間の同期をとれます。
  //同期を取るということは、全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];usum[tid]+=usum[tid+64];} //__syncthreads();は複数個必要1個だけ記述したら数が違った
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];usum[tid]+=usum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];usum[tid]+=usum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];usum[tid]+=usum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];usum[tid]+=usum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];usum[tid]+=usum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];usum[tid]+=usum[tid+1];} 
  __syncthreads();if(tid==0){d_total[bid]=sum[0];d_uniq[bid]=usum[0];}
  //__syncthreads();//これだとn13以降数が合わない
  //for (int k = 0; k < THREAD_NUM; ++k){
  //  d_total[bid]+=sum[k];
  //  d_uniq[bid]+=usum[k];
  //}
  //__syncthreads();

}
//
// GPU
void backTrack1G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps) {//NQueenに相当
  register int bitmap[32];//bitmapを配列で持つことによりstackを使わないで1行前に戻れる
  register int bit;

  //host
  register int h_down[size];
  hipHostMalloc((void**) &h_down,sizeof(int)*steps, hipHostMallocDefault);
  register int h_right[size];
  hipHostMalloc((void**) &h_right,sizeof(int)*steps, hipHostMallocDefault);
  register int h_left[size];
  hipHostMalloc((void**) &h_left,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_down=new unsigned int[steps];
  hipHostMalloc((void**) &t_down,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_left=new unsigned int[steps];
  hipHostMalloc((void**) &t_left,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_right=new unsigned int[steps];
  hipHostMalloc((void**) &t_right,sizeof(int)*steps, hipHostMallocDefault);
  //unsigned int t_aBoard[steps][MAX];
  unsigned int* t_aBoard=new unsigned int[steps*MAX];
  hipHostMalloc((void**) &t_aBoard,sizeof(int)*steps*MAX, hipHostMallocDefault);
  unsigned int* h_total=new unsigned int[steps];
  hipHostMalloc((void**) &h_total,sizeof(int)*steps/THREAD_NUM, hipHostMallocDefault);
  unsigned int* h_uniq=new unsigned int[steps];
  hipHostMalloc((void**) &h_uniq,sizeof(int)*steps/THREAD_NUM, hipHostMallocDefault);

  //device
  unsigned int* d_down;
  hipMalloc((void**) &d_down,sizeof(int)*steps);
  unsigned int* d_left;
  hipMalloc((void**) &d_left,sizeof(int)*steps);
  unsigned int* d_right;
  hipMalloc((void**) &d_right,sizeof(int)*steps);
  unsigned int* d_total;
  hipMalloc((void**) &d_total,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  //int** d_aBoard;//GPU内で２次元配列として使いたい場合
  unsigned int* d_aBoard;
  hipMalloc((void**) &d_aBoard,sizeof(int)*steps*MAX);

  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  const unsigned int mark=size>11?size-9:3;
  const unsigned int h_mark=row;

  h_left[row]=n_left;
  h_down[row]=n_down;
  h_right[row]=n_right;
  bitmap[row]=mask&~(h_left[row]|h_down[row]|h_right[row]);

  //12行目までは3行目までCPU->row==mark以下で 3行目までのdown,left,right情報を t_down,t_left,t_rightに格納する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く　例えば n15だとrow=5までCPUで実行し、それ以降はGPU(現在の設定だとGPUでは最大10行実行するようになっている)
  int totalCond=0;
  //bit=0;
  //h_down[0]=h_left[0]=h_right[0]=0;
  bool matched=false;
  while(row>=h_mark){
    if(bitmap[row]==0){//bitmap[row]=00000000 クイーンをどこにも置けないので1行上に戻る
        row--;
    }else{//おける場所があれば進む
        if(row<BOUND1) {
          bitmap[row]&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
        }
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); //クイーンを置く
        if((bit&mask)!=0){//置く場所があれば先に進む
          int n=row++;//クイーン置いた位置から次の行へ渡す down,left,right,bitmapを出す
          h_down[row]=h_down[n]|bit;
          h_left[row]=(h_left[n]|bit)<<1;
          h_right[row]=(h_right[n]|bit)>>1;
          bitmap[row]=mask&~(h_down[row]|h_left[row]|h_right[row]);
          if(row==mark){
            //3行目(mark)にクイーンを１個ずつ置いていって、down,left,right情報を格納、
            //その次の行へは進まない。その行で可能な場所にクイーン置き終わったらGPU並列実行
            t_down[totalCond]=h_down[row];//totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
            t_left[totalCond]=h_left[row];//row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報をt_down,t_left,t_rightに格納する
            t_right[totalCond]=h_right[row];
            for(int i=0;i<size;i++){
              //t_aBoard[totalCond][i]=aBoard[i];
              t_aBoard[totalCond*MAX+i]=aBoard[i];

            }
            totalCond++;//スレッド数をインクリメントする
            //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同時並行稼働数を制御
            //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増えて行くと超えるようになる。
            if(totalCond==steps){//ここではtotalCond==stepsの場合だけこの中へ
              if(matched){//matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
                hipMemcpy(h_total,d_total,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                hipMemcpy(h_uniq,d_uniq,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){TOTAL+=h_total[col];UNIQUE+=h_uniq[col];}
                matched=false;
              }
              hipMemcpy(d_down,t_down,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(d_left,t_left,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(d_right,t_right,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(d_aBoard,t_aBoard,
                  sizeof(int)*totalCond*MAX,hipMemcpyHostToDevice);
              //cudaMemcpy(d_aBoard,t_aBoard,
              //    sizeof(int)*totalCond*MAX,cudaMemcpyHostToDevice);
              //cudaMemcpyToSymbol(d_aBoard,t_aBoard,
              //    sizeof(int)*totalCond*MAX,0);
              
              /** backTrack+bitmap*/
              cuda_kernel_b1<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,d_down,d_left,d_right,d_total,d_uniq,d_aBoard,totalCond,row,BOUND1);//size-mark は何行GPUを実行するか totalCondはスレッド数
              //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのはtotalCondの数だけでそれ以外は空回しになる
              matched=true;//GPU内でカウントしているので、GPUから出たらmatched=trueになってる
              totalCond=0;//totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始する(これによりなんどもsteps数分だけGPUを起動できる)
              
            }
            --row;//t_down,t_left,t_rightに情報を格納したら1行上に上がる
            //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いてt_down,t_left,t_rightに情報を格納する
          }
        }else{//置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通にnqueenをやる
          --row;
        }
      }
    }
   if(matched){//matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
               hipMemcpy(h_total,d_total,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
               hipMemcpy(h_uniq,d_uniq,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
               for(int col=0;col<steps/THREAD_NUM;col++){TOTAL+=h_total[col];UNIQUE+=h_uniq[col];}
               matched=false;}
               hipMemcpy(d_down,t_down,sizeof(int)*totalCond,hipMemcpyHostToDevice);
hipMemcpy(d_left,t_left,sizeof(int)*totalCond,hipMemcpyHostToDevice);
hipMemcpy(d_right,t_right,sizeof(int)*totalCond,hipMemcpyHostToDevice);
hipMemcpy(d_aBoard,t_aBoard,sizeof(int)*totalCond*MAX,hipMemcpyHostToDevice);
//cudaMemcpyToSymbol(d_aBoard,t_aBoard,//２次元配列で使いたい場合
//                  sizeof(int)*totalCond*MAX,0);
                            
    /** backTrack+bitmap*/            
   cuda_kernel_b1<<<steps/THREAD_NUM,THREAD_NUM
   >>>(size,size-mark,d_down,d_left,d_right,d_total,d_uniq,d_aBoard,totalCond,mark,BOUND1);//size-mark は何行GPUを実行するか totalCondはスレッド数
   //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのはtotalCondの数だけでそれ以外は空回しになる
    hipMemcpy(h_total,d_total,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    hipMemcpy(h_uniq,d_uniq,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    
    for(int col=0;col<steps/THREAD_NUM;col++){TOTAL+=h_total[col];UNIQUE+=h_uniq[col];}
    
   hipFree(d_down);
   hipFree(d_left);
   hipFree(d_right);
   hipFree(d_total);
   hipFree(d_uniq);
   hipFree(d_aBoard);
   hipHostFree(t_down);
   hipHostFree(t_left);
   hipHostFree(t_right);
   hipHostFree(t_aBoard);
   hipHostFree(h_down);
   hipHostFree(h_left);
   hipHostFree(h_right);
   hipHostFree(h_total);
   hipHostFree(h_uniq);
}
//
void backTrack2G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps) {//NQueenに相当
  register int bitmap[32];//bitmapを配列で持つことによりstackを使わないで1行前に戻れる
  register int bit;

  //host
  register int h_down[size];
  hipHostMalloc((void**) &h_down,sizeof(int)*steps, hipHostMallocDefault);
  register int h_right[size];
  hipHostMalloc((void**) &h_right,sizeof(int)*steps, hipHostMallocDefault);
  register int h_left[size];
  hipHostMalloc((void**) &h_left,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_down=new unsigned int[steps];
  hipHostMalloc((void**) &t_down,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_left=new unsigned int[steps];
  hipHostMalloc((void**) &t_left,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_right=new unsigned int[steps];
  hipHostMalloc((void**) &t_right,sizeof(int)*steps, hipHostMallocDefault);
  //unsigned int t_aBoard[steps][MAX];
  unsigned int* t_aBoard=new unsigned int[steps*MAX];
  hipHostMalloc((void**) &t_aBoard,sizeof(int)*steps*MAX, hipHostMallocDefault);
  unsigned int* h_total=new unsigned int[steps];
  hipHostMalloc((void**) &h_total,sizeof(int)*steps/THREAD_NUM, hipHostMallocDefault);
  unsigned int* h_uniq=new unsigned int[steps];
  hipHostMalloc((void**) &h_uniq,sizeof(int)*steps/THREAD_NUM, hipHostMallocDefault);

  //device
  unsigned int* d_down;
  hipMalloc((void**) &d_down,sizeof(int)*steps);
  unsigned int* d_left;
  hipMalloc((void**) &d_left,sizeof(int)*steps);
  unsigned int* d_right;
  hipMalloc((void**) &d_right,sizeof(int)*steps);
  unsigned int* d_total;
  hipMalloc((void**) &d_total,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  //int** d_aBoard;//GPU内で２次元配列として使いたい場合
  unsigned int* d_aBoard;
  hipMalloc((void**) &d_aBoard,sizeof(int)*steps*MAX);

  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  unsigned int mark=size>11?size-9:3;
  if(size<8){
   mark=2;   
  }
  const unsigned int h_mark=row;

  h_left[row]=n_left;
  h_down[row]=n_down;
  h_right[row]=n_right;
  bitmap[row]=mask&~(h_left[row]|h_down[row]|h_right[row]);

  //12行目までは3行目までCPU->row==mark以下で 3行目までのdown,left,right情報を t_down,t_left,t_rightに格納する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く　例えば n15だとrow=5までCPUで実行し、それ以降はGPU(現在の設定だとGPUでは最大10行実行するようになっている)
  int totalCond=0;
  //bit=0;
  //h_down[0]=h_left[0]=h_right[0]=0;
  bool matched=false;
  while(row>=h_mark){
    if(bitmap[row]==0){//bitmap[row]=00000000 クイーンをどこにも置けないので1行上に戻る
        row--;
        }else{//おける場所があれば進む
        //【枝刈り】上部サイド枝刈り
	      if(row<BOUND1){     
          //printf("logic_row:%d:b1:%d\n",row,BOUND1);        	
	        bitmap[row]&=~SIDEMASK;
        //【枝刈り】下部サイド枝刈り
        }else if(row==BOUND2) {     	
          //printf("logic_row:%d:BOUND2:%d\n",row,BOUND2);
          if((down[row]&SIDEMASK)==0){ row--; }
          if((down[row]&SIDEMASK)!=SIDEMASK){ bitmap[row]&=SIDEMASK; }
        }

        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); //クイーンを置く
        if((bit&mask)!=0){//置く場所があれば先に進む
          int n=row++;//クイーン置いた位置から次の行へ渡す down,left,right,bitmapを出す
          h_down[row]=h_down[n]|bit;
          h_left[row]=(h_left[n]|bit)<<1;
          h_right[row]=(h_right[n]|bit)>>1;
          bitmap[row]=mask&~(h_down[row]|h_left[row]|h_right[row]);
          if(row==mark){
            //3行目(mark)にクイーンを１個ずつ置いていって、down,left,right情報を格納、
            //その次の行へは進まない。その行で可能な場所にクイーン置き終わったらGPU並列実行
            t_down[totalCond]=h_down[row];//totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
            t_left[totalCond]=h_left[row];//row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報をt_down,t_left,t_rightに格納する
            t_right[totalCond]=h_right[row];
            //printf("down:%d:left:%d:right:%d\n",t_down[totalCond],t_left[totalCond],t_right[totalCond]);
            for(int i=0;i<size;i++){
              //t_aBoard[totalCond][i]=aBoard[i];
              t_aBoard[totalCond*MAX+i]=aBoard[i];

            }
            totalCond++;//スレッド数をインクリメントする
            //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同時並行稼働数を制御
            //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増えて行くと超えるようになる。
            if(totalCond==steps){//ここではtotalCond==stepsの場合だけこの中へ
              if(matched){//matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
                hipMemcpy(h_total,d_total,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                hipMemcpy(h_uniq,d_uniq,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){TOTAL+=h_total[col];UNIQUE+=h_uniq[col];}
                matched=false;
              }
              hipMemcpy(d_down,t_down,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(d_left,t_left,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(d_right,t_right,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(d_aBoard,t_aBoard,
                  sizeof(int)*totalCond*MAX,hipMemcpyHostToDevice);
              //cudaMemcpy(d_aBoard,t_aBoard,
              //    sizeof(int)*totalCond*MAX,cudaMemcpyHostToDevice);
              //cudaMemcpyToSymbol(d_aBoard,t_aBoard,
              //    sizeof(int)*totalCond*MAX,0);
              
              /** backTrack+bitmap*/
              cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,d_down,d_left,d_right,d_total,d_uniq,d_aBoard,totalCond,row,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT);//size-mark は何行GPUを実行するか totalCondはスレッド数
              //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのはtotalCondの数だけでそれ以外は空回しになる
              matched=true;//GPU内でカウントしているので、GPUから出たらmatched=trueになってる
              totalCond=0;//totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始する(これによりなんどもsteps数分だけGPUを起動できる)
              
            }
            --row;//t_down,t_left,t_rightに情報を格納したら1行上に上がる
            //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いてt_down,t_left,t_rightに情報を格納する
          }
        }else{//置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通にnqueenをやる
          --row;
        }
      }
    }
   if(matched){//matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
               hipMemcpy(h_total,d_total,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
               hipMemcpy(h_uniq,d_uniq,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
               for(int col=0;col<steps/THREAD_NUM;col++){TOTAL+=h_total[col];UNIQUE+=h_uniq[col];}
               matched=false;}
               hipMemcpy(d_down,t_down,sizeof(int)*totalCond,hipMemcpyHostToDevice);
hipMemcpy(d_left,t_left,sizeof(int)*totalCond,hipMemcpyHostToDevice);
hipMemcpy(d_right,t_right,sizeof(int)*totalCond,hipMemcpyHostToDevice);
hipMemcpy(d_aBoard,t_aBoard,sizeof(int)*totalCond*MAX,hipMemcpyHostToDevice);
//cudaMemcpyToSymbol(d_aBoard,t_aBoard,//２次元配列で使いたい場合
//                  sizeof(int)*totalCond*MAX,0);
                            
    /** backTrack+bitmap*/          
   //printf("tc:%d:mark:%d:b1:%d:b2:%d:sm:%d:lm:%d:tb:%d:eb:%d\n",totalCond,mark,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT);  
   cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
   >>>(size,size-mark,d_down,d_left,d_right,d_total,d_uniq,d_aBoard,totalCond,mark,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT);//size-mark は何行GPUを実行するか totalCondはスレッド数
   //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのはtotalCondの数だけでそれ以外は空回しになる
    hipMemcpy(h_total,d_total,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    hipMemcpy(h_uniq,d_uniq,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    
    for(int col=0;col<steps/THREAD_NUM;col++){TOTAL+=h_total[col];UNIQUE+=h_uniq[col];}
    
   hipFree(d_down);
   hipFree(d_left);
   hipFree(d_right);
   hipFree(d_total);
   hipFree(d_uniq);
   hipFree(d_aBoard);
   hipHostFree(t_down);
   hipHostFree(t_left);
   hipHostFree(t_right);
   hipHostFree(t_aBoard);
   hipHostFree(h_down);
   hipHostFree(h_left);
   hipHostFree(h_right);
   hipHostFree(h_total);
   hipHostFree(h_uniq);
}

void NQueenG(int size,int steps){
  int bit=0;
  int mask=(1<<size)-1;
  TOPBIT=1<<(size-1);
  //11から枝借りをする
  //backtrack1
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      BOUND1=col_j;
      backTrack1G(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1,steps);
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  //printf("size:%d:Sidemask:%d\n",size,SIDEMASK);
  ENDBIT=(TOPBIT>>1);
  //backtrack2
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      BOUND1=col;
      BOUND2=col2;
      backTrack2G(size,mask,1,bit<<1,bit,bit>>1,steps);
      LASTMASK|=LASTMASK>>1|LASTMASK<<1;
      ENDBIT>>=1;
  }
}

//CPU 非再帰版 ロジックメソッド
void backTrack1(int size,int mask, int row,int h_left,int h_down,int h_right){
    unsigned int left[size];
    unsigned int down[size];
    unsigned int right[size];
    unsigned int bitmap[size];
    left[row]=h_left;
    down[row]=h_down;
    right[row]=h_right;
    bitmap[row]=mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        if(row<BOUND1) {
          bitmap[row]&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
        }
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            //symmetryOps_bitmap(size);
            COUNT8++;
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
void backTrack2(int size,int mask, int row,int h_left,int h_down,int h_right){
    unsigned int left[size];
    unsigned int down[size];
    unsigned int right[size];
    unsigned int bitmap[size];
    left[row]=h_left;
    down[row]=h_down;
    right[row]=h_right;
    bitmap[row]=mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
	      //【枝刈り】上部サイド枝刈り
	      if(row<BOUND1){             	
	        //printf("BOUND1_row:%d:bit:%d\n",row,bitmap[row]);
          bitmap[row]&=~SIDEMASK;
        
        //【枝刈り】下部サイド枝刈り
        }else if(row==BOUND2) {     	
	        //printf("BOUND2_row:%d:bit:%d\n",row,bitmap[row]);
          if((down[row]&SIDEMASK)==0){ 
              //printf("BOUND2_row\n");
              row--; 
          }
          if((down[row]&SIDEMASK)!=SIDEMASK){ 
              //printf("BOUND2_row_SIDE\n");
              bitmap[row]&=SIDEMASK;
              }
        }
        int save_bitmap=bitmap[row];
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            if((save_bitmap&LASTMASK)==0){ 	
              symmetryOps(size);
              --row;
		    }
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
void NQueen(int size,int mask){
  int bit=0;
  TOPBIT=1<<(size-1);
  //11から枝借りをする
  //backtrack1
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      BOUND1=col_j;
      backTrack1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1);
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  //backtrack2
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      BOUND1=col;
      BOUND2=col2;
      backTrack2(size,mask,1,bit<<1,bit,bit>>1);
      LASTMASK|=LASTMASK>>1|LASTMASK<<1;
      ENDBIT>>=1;
  }
}
//
//CPUR 再帰版 ロジックメソッド
void backTrackR1(int size,int mask, int row,int left,int down,int right){
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
   if(bitmap){
     COUNT8++;
   }
  }else{
    if(row<BOUND1) {
      bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
    }
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrackR1(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
void backTrackR2(int size,int mask, int row,int left,int down,int right){
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
   if(bitmap){
     //【枝刈り】 最下段枝刈り
     if((bitmap&LASTMASK)==0){ 	
       aBoard[row]=(-bitmap&bitmap);
       symmetryOps(size);
     }
   }
  }else{
    //【枝刈り】上部サイド枝刈り
    if(row<BOUND1){             	
      bitmap&=~SIDEMASK;
      //【枝刈り】下部サイド枝刈り
    }else if(row==BOUND2) {     	
      if((down&SIDEMASK)==0){ return; }
      if((down&SIDEMASK)!=SIDEMASK){ bitmap&=SIDEMASK; }
    }
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrackR2(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//
//CPUR 再帰版 ロジックメソッド
void NQueenR(int size,int mask){
  int bit=0;
  TOPBIT=1<<(size-1);
  //11から枝借りする
  //backtrack1
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      BOUND1=col_j;
      backTrackR1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1);
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  //backtrack2
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      BOUND1=col;
      BOUND2=col2;
      backTrackR2(size,mask,1,bit<<1,bit,bit>>1);
      LASTMASK|=LASTMASK>>1|LASTMASK<<1;
      ENDBIT>>=1;
  }
}
//
//CPU 非再帰版 backTrack2
void backTrack2D_NR(int size,int mask,int row,int left,int down,int right){
	int bitmap,bit;
	int b[100], *p=b;
  int sizeE=size-1;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
    mais1:bitmap=mask&~(left|down|right);
    // 【枝刈り】
    //if(row==size){
    if(row==sizeE){
      //if(!bitmap){
      if(bitmap){
        //【枝刈り】 最下段枝刈り
        if((bitmap&LASTMASK)==0){
          aBoard[row]=bitmap; //symmetryOpsの時は代入します。
          symmetryOps(size);
        }
      }
    }else{
      //【枝刈り】上部サイド枝刈り
      if(row<BOUND1){
        bitmap&=~SIDEMASK;
        //【枝刈り】下部サイド枝刈り
      }else if(row==BOUND2){
        if(!(down&SIDEMASK))
          goto volta;
        if((down&SIDEMASK)!=SIDEMASK)
          bitmap&=SIDEMASK;
      }
      if(bitmap){
        outro:bitmap^=aBoard[row]=bit=-bitmap&bitmap;
        if(bitmap){
          *p++=left;
          *p++=down;
          *p++=right;
        }
        *p++=bitmap;
        row++;
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        goto mais1;
        //Backtrack2(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
        volta:if(p<=b)
          return;
        row--;
        bitmap=*--p;
        if(bitmap){
          right=*--p;
          down=*--p;
          left=*--p;
          goto outro;
        }else{
          goto volta;
        }
      }
    }
    goto volta;
  }
}
//CPU 非再帰版 backTrack
void backTrack1D_NR(int size,int mask,int row,int left,int down,int right){
	int bitmap,bit;
	int b[100], *p=b;
  int sizeE=size-1;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
    b1mais1:bitmap=mask&~(left|down|right);
    //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
    //if(row==size){
    if(row==sizeE){
      //if(!bitmap){
      if(bitmap){
        // aBoard[row]=bitmap;
        //symmetryOps(size);
        COUNT8++;
      }
    }else{
      //【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい
      // ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
      if(row<BOUND1) {
        bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
      }
      if(bitmap){
  b1outro:bitmap^=aBoard[row]=bit=-bitmap&bitmap;
        if(bitmap){
          *p++=left;
          *p++=down;
          *p++=right;
        }
        *p++=bitmap;
        row++;
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        goto b1mais1;
        //Backtrack1(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
  b1volta:if(p<=b)
          return;
        row--;
        bitmap=*--p;
        if(bitmap){
          right=*--p;
          down=*--p;
          left=*--p;
          goto b1outro;
        }else{
          goto b1volta;
        }
      }
    }
    goto b1volta;
  }
}
//CPU 非再帰版 ロジックメソッド
void NQueenD(int size,int mask){
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    //backTrack1(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
    backTrack1D_NR(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    //backTrack1(size,mask,1,bit<<1,bit,bit>>1);
    backTrack2D_NR(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
//
void backTrack2D(int size,int mask,int row,int left,int down,int right){
  int bit;
  int bitmap=mask&~(left|down|right);
  // 【枝刈り】
  if(row==size-1){ 								
    if(bitmap){
      //【枝刈り】 最下段枝刈り
      if((bitmap&LASTMASK)==0){ 	
        aBoard[row]=bitmap; //symmetryOpsの時は代入します。
        symmetryOps(size);
      }
    }
  }else{
    //【枝刈り】上部サイド枝刈り
    if(row<BOUND1){             	
      bitmap&=~SIDEMASK;
      //【枝刈り】下部サイド枝刈り
    }else if(row==BOUND2) {     	
      if((down&SIDEMASK)==0){ return; }
      if((down&SIDEMASK)!=SIDEMASK){ bitmap&=SIDEMASK; }
    }
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrack2D(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
//
void backTrack1D(int size,int mask,int row,int left,int down,int right){
  int bit;
  int bitmap=mask&~(left|down|right);
  //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
  if(row==size-1) {
    if(bitmap){
      // aBoard[row]=bitmap;
      COUNT8++;
    }
  }else{
    //【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい
    // ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
    if(row<BOUND1) {
      bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
    }
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrack1D(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
//
//CPUR 再帰版 ロジックメソッド
void NQueenDR(int size,int mask){
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    backTrack1D(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    backTrack2D(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
//
//メインメソッド
int main(int argc,char** argv) {
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */  
  if(cpu){
    printf("\n\n１２．CPU 非再帰 枝刈り\n");
  }else if(cpur){
    printf("\n\n１２．CPUR 再帰 枝刈り\n");
  }else if(gpu){
    printf("\n\n１２．GPU 非再帰 枝刈り\n");
  }else if(sgpu){
    printf("\n\n１２．SGPU 非再帰 バックトラック＋ビットマップ\n");
  }

  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;           //速度計測用
    char t[20];           //hh:mm:ss.msを格納
    int min=4; int targetN=17;
    
    int mask;
    for(int i=min;i<=targetN;i++){
      //TOTAL=0; UNIQUE=0;
      COUNT2=COUNT4=COUNT8=0;
      mask=(1<<i)-1;
      st=clock();
      //初期化は不要です
      /** 非再帰は-1で初期化 */
      // for(int j=0;j<=targetN;j++){
      //   aBoard[j]=-1;
      // }
      //
      //再帰
      if(cpur){ 
        NQueenR(i,mask);
        //printf("通常版\n");
        //NQueenDR(i,mask);//通常版
      }
      //非再帰
      if(cpu){ 
        NQueen(i,mask); 
        //printf("通常版\n");
        //NQueenD(i,mask);//通常版
      }
      //
      TimeFormat(clock()-st,t); 
      printf("%2d:%13ld%16ld%s\n",i,getTotal(),getUnique(),t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=17;
    struct timeval t0;struct timeval t1;int ss;int ms;int dd;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      TOTAL=0;
      UNIQUE=0;
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        NQueenG(i,steps);
      }else if(sgpu){
        TOTAL=sgpu_solve_nqueen_cuda(i,steps);
      }
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
